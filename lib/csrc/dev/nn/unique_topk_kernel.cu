#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cuda/std/type_traits>
#include <cstdio>
#include "shared_nn_utils.cu"

using namespace at;
// #define AT_DISPATCH_CASE_ITYPE(...)   \
//   AT_DISPATCH_CASE(at::ScalarType::Int32, __VA_ARGS__) \
//   AT_DISPATCH_CASE(at::ScalarType::Double, __VA_ARGS__) \
//   AT_DISPATCH_CASE(at::ScalarType::Float, __VA_ARGS__)  \
//   AT_DISPATCH_CASE(at::ScalarType::Half, __VA_ARGS__)
// #define AT_DISPATCH_ITYPES(TYPE, NAME, ...) \
//   AT_DISPATCH_SWITCH(TYPE, NAME, AT_DISPATCH_CASE_ITYPE(__VA_ARGS__))
// at::ScalarType get_type(torch::Tensor my_tensor);

/****************************

       Forward Pass

****************************/

template <typename scalar_t, typename itype>
__global__ void unique_topk_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<itype,3,torch::RestrictPtrTraits> inds,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> dists_topk,
    torch::PackedTensorAccessor32<itype,3,torch::RestrictPtrTraits> inds_topk,
    int k, int qpt){

  // -- shaping vars --
  int qi_cuda = threadIdx.x + blockDim.x * blockIdx.x;
  int Q = dists.size(0);
  int k_in = dists.size(1);

  // -- major index --
  int kj = 0;
  int qi = 0;

  // -- checking vars --
  int kj0 = 0;
  int diff;
  int kl;
  scalar_t dist_curr,dist_prev;
  bool any_same,reset_check;
  int check_start;
  dist_prev = (scalar_t)__int_as_float(0x7f800000);

  // -- iterate over blocks --
  for(int qi_ix = 0; qi_ix < qpt; qi_ix++){

    // -- specify query index --
    qi = qi_cuda + qi_ix;
    if (qi >= Q){ continue; }

    // -- init --
    kj0 = 0;
    check_start = 0;

    // -- find value for each "k" location --
    for (int ki = 0; ki < k; ki++){

      // -- terminate (& fail) if kj,ki too beyond limits --
      if (kj0 >= k_in){
        break;
      }

      // -- assign --
      dist_curr = dists[qi][kj0];
      dists_topk[qi][ki] = dist_curr;
      inds_topk[qi][ki][0] = inds[qi][kj0][0];
      inds_topk[qi][ki][1] = inds[qi][kj0][1];
      inds_topk[qi][ki][2] = inds[qi][kj0][2];

      // -- update "same dist" history --
      reset_check = fabs((float)(dist_curr - dist_prev))>1e-5;
      check_start = reset_check ? kj0 : check_start;
      dist_prev = dist_curr;

      // -- find next kj --
      for (kj = kj0+1; kj < k_in; kj++){

        // -- Keep moving "kj" forward until we find one that is different
        // -- than all previous "kj" indices with the same "dist" value.
        any_same = false;
        for (kl = check_start; kl <= kj0; kl++){

          diff = 0;
          #pragma unroll
          for (int ix = 0; ix < 3; ix++){
            if (is_same_v<itype,int>){
            // if (itype == int){
              diff += inds[qi][kj][ix] != inds[qi][kl][ix];
            }else{
              diff += fabs(inds[qi][kj][ix] - inds[qi][kl][ix]) > 1e-8;
            }
          }
          any_same = (diff == 0) || any_same;
        }

        // -- stop incrementing at unique kj --
        if (not any_same){
          break;
        }          

      }
      kj0 = kj; // update selected neighbor

    }
  }
}

void unique_topk_forward_cuda(const torch::Tensor dists, const torch::Tensor inds,
                              torch::Tensor dists_topk, torch::Tensor inds_topk,
                              int k){

   // -- comp per threads --
   int Q = dists.size(0);
   int K = dists.size(1);

   // -- blocks --
   int queries_per_thread = 1;
   int nthreads = 256;
   int nblocks = (Q-1)/(queries_per_thread*nthreads)+1;

   // launch kernel
   // const auto& itype_str = inds.type();
   // at::ScalarType _st = ::detail::scalar_type(the_type);
   auto itype = get_type(inds);
   auto dtype = get_type(dists);
   if (itype == torch::kInt32){
     AT_DISPATCH_FLOATING_TYPES(dists.type(),"unique_topk_forward_kernel", ([&] {
         unique_topk_forward_kernel<scalar_t,int><<<nblocks, nthreads>>>(
          dists.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,3,torch::RestrictPtrTraits>(),
          dists_topk.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
          inds_topk.packed_accessor32<int,3,torch::RestrictPtrTraits>(),
          k, queries_per_thread);
        }));
   }else if (itype == dtype){
     AT_DISPATCH_FLOATING_TYPES(dists.type(),"unique_topk_forward_kernel", ([&] {
         unique_topk_forward_kernel<scalar_t,scalar_t><<<nblocks, nthreads>>>(
          dists.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
          dists_topk.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
          inds_topk.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
          k, queries_per_thread);
        }));
   }else{
     std::cout << "Must have inds type be int or match dists.type().\n"<< std::endl;
     assert(0==1);
   }
}

// at::ScalarType get_type(torch::Tensor my_tensor){
//   const auto& the_type = my_tensor.type();
//   at::ScalarType _st = ::detail::scalar_type(the_type);
//   return _st;
// }

/****************************

       Backward Pass

****************************/

// none yet