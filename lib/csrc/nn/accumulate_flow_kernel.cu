#include "hip/hip_runtime.h"

#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <assert.h>


#include <cstddef>
#include <math.h>
#include <ATen/ATen.h>
#include <cuda/std/type_traits>
#include <cstdio>
#include "../search/nls_bilin2d.cu"
// #include "shared_nn_utils.cu"
// #include "../search/nls_bilin2d.cu"


using namespace at;

// template<typename dtype=int>
// __device__ __forceinline__ dtype bounds_clip(dtype val, int lim ){
//   dtype vval = val;
//   if (val < 0){
//     vval = -val; // want ("-1" -> "1") _not_ ("-1" -> "0")
//     vval = vval > (lim-1) ? 0 : vval;
//   }else if (val > (lim-1)){
//     vval = 2*(lim-1)-val; // want ("H" -> "H-2") _not_ ("H" -> "H-1")
//     vval = vval < 0 ? lim-1 : vval;
//   }
//   return vval;
// }

// template<typename scalar_t>
// __device__ __forceinline__ 
// void bilinear_index(
//      const torch::TensorAccessor<scalar_t,3,torch::RestrictPtrTraits,int32_t> flow,
//      scalar_t& v0, scalar_t& v1,
//      scalar_t hj_center, scalar_t wj_center, int H, int W){
//   v0=0;
//   v1=0;
// #pragma unroll
//     for (int i=0;i<2;i++){
// #pragma unroll
//       for (int j=0;j<2;j++){

//         // -- compute int locaion with weight --
//         hj = __float2int_rd(hj_center + i);
//         wj = __float2int_rd(wj_center + j);
//         weight = max(0.,1-fabs(hj-hj_center)) * max(0.,1-fabs(wj-wj_center));

//         // -- ensure legal boudns --
//         hj = bounds(hj,H);
//         wj = bounds(wj,W);

//         // -- update with shift --
//         v0 += weight*flow[0][hj][wj];
//         v1 += weight*flow[1][hj][wj];
//       }
//     }
// }

template<typename scalar_t, typename itype=int>
__device__ __forceinline__ 
void update_centers_flow_acc(itype& hj_center, itype& wj_center, int H, int W,
  const torch::TensorAccessor<scalar_t,3,torch::RestrictPtrTraits,int32_t> flow){


  // -- fixed so we can read both --
  itype hj_tmp = hj_center;
  itype wj_tmp = wj_center;

  // -- update --
  if(is_same_v<itype,int>){

    // // -- simple rounding if "int" --
    // wj_center = int(1.*wj_center + flow[0][hj_tmp][wj_tmp] + 0.5);
    // hj_center = int(1.*hj_center + flow[1][hj_tmp][wj_tmp] + 0.5);

    // // -- wrap around boarders --
    // wj_center = max(0,min(W-1,(int)wj_center));
    // hj_center = max(0,min(H-1,(int)hj_center));

  }else{


    // -- weighted average of neighbors --
    float weight = 0;
    int hj = 0, wj = 0;
#pragma unroll
    for (int i=0;i<2;i++){
#pragma unroll
      for (int j=0;j<2;j++){

        // -- compute int locaion with weight --
        hj = __float2int_rd(hj_tmp + i);
        wj = __float2int_rd(wj_tmp + j);
        weight = max(0.,1-fabs(hj-hj_tmp)) * max(0.,1-fabs(wj-wj_tmp));

        // -- ensure legal boudns --
        hj = bounds(hj,H);
        wj = bounds(wj,W);

        // -- update with shift --
        wj_center += weight*flow[0][hj][wj];
        hj_center += weight*flow[1][hj][wj];
      }
    }

    // -- wrap around boarders --
    // wj_center = max((float)0.,(float)min((float)1.*W-1,(float)wj_center));
    // hj_center = max((float)0.,(float)min((float)1.*H-1,(float)hj_center));

  }
}

template <typename scalar_t, typename itype>
__global__ void accumulate_flow_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fflow,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> bflow,
    torch::PackedTensorAccessor32<itype,6,torch::RestrictPtrTraits> pfflow,
    torch::PackedTensorAccessor32<itype,6,torch::RestrictPtrTraits> pbflow,
    int stride0, int locs_per_thread){

  // -- unpack --
  int bi = blockIdx.y;
  int raster_index = locs_per_thread*(threadIdx.x + blockDim.x * blockIdx.x);
  int T = fflow.size(1);
  int H = fflow.size(3);
  int W = fflow.size(4);
  int nH = (H-1)/stride0+1;
  int nW = (W-1)/stride0+1;
  int nHW = nH*nW;
  int TnHW = T*nH*nW;
  int tmp;
  int ref[3];

  // -- get location --
  for (int loc = 0; loc < locs_per_thread; loc++){

    // -- get location --
    int qi = raster_index + loc;
    if (qi >= TnHW){ return; } 
    get_pixel_loc(ref,qi,tmp,stride0,nW,nHW,H,W);
    int ti = ref[0];
    int wn = ref[1];
    int hn = ref[2];

    itype hi_a,wi_a;
    if (is_same_v<itype,int>){
      hi_a = (hn * stride0) % H;
      wi_a = (wn * stride0) % W;
    }else{
      hi_a = trunc(__int2float_rn((hn * stride0) % H));
      wi_a = trunc(__int2float_rn((wn * stride0) % W));
    }

    // -- run left --
    int ta = 0;
    auto flow = bflow;
    auto pflow = pbflow;
    itype h_center = hi_a;
    itype w_center = wi_a;
    for(int tj=ti; tj > 0; tj--){

      // -- accumulate center offset  --
      update_centers_flow_acc<scalar_t,itype>(h_center,w_center,H,W,flow[bi][tj]);

      // -- assignment  --
      pflow[bi][ti][ta][1][hn][wn] = h_center - hi_a;
      pflow[bi][ti][ta][0][hn][wn] = w_center - wi_a;

      // -- incriment pre-computed frame index --
      ta++;
    }

    // -- run right --
    ta = 0;
    flow = fflow;
    pflow = pfflow;
    h_center = hi_a;
    w_center = wi_a;
    for(int tj=ti; tj < (T-1); tj++){

      // -- accumulate center offset  --
      update_centers_flow_acc(h_center,w_center,H,W,flow[bi][tj]);

      // -- assignment  --
      // pflow[bi][ti][ta][1][hn][wn] = h_center - hi_a;
      // pflow[bi][ti][ta][0][hn][wn] = w_center - wi_a;
      pflow[bi][ti][ta][1][hn][wn] = h_center - hi_a;
      pflow[bi][ti][ta][0][hn][wn] = w_center - wi_a;

      // -- incriment pre-computed frame index --
      ta++;

    }
  }
    
}


void accumulate_flow_forward_cuda(
     const torch::Tensor fflow, const torch::Tensor bflow,
     torch::Tensor pfflow, torch::Tensor pbflow, int stride0){
  
  // -- unpack --
  int B = fflow.size(0);
  int T = fflow.size(1);
  int H = fflow.size(3);
  int W = fflow.size(4);

  // -- num 2 run --
  int nH = (H-1)/stride0+1;
  int nW = (W-1)/stride0+1;
  int nRun = T*nH*nW;

  // -- kernel params --
  int locs_per_thread = 1;
  int _nthreads = 256;
  dim3 nthreads(_nthreads);
  int _nblocks = (nRun-1)/(_nthreads*locs_per_thread)+1;
  dim3 nblocks(_nblocks,B);
  // fprintf(stdout,"nblocks,nthreads: %d,%d\n",_nblocks,_nthreads);
  // fprintf(stdout,"stride0: %d\n",stride0);

  // -- launch kernel --
  if(pfflow.dtype() == torch::kInt32){
    AT_DISPATCH_FLOATING_TYPES(fflow.type(), "accumulate_flow_forward_kernel", ([&] {
        accumulate_flow_forward_kernel<scalar_t,int><<<nblocks, nthreads>>>(
         fflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
         bflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
         pfflow.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
         pbflow.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
         stride0,locs_per_thread);
        }));
  }else{
    AT_DISPATCH_FLOATING_TYPES(fflow.type(), "accumulate_flow_forward_kernel", ([&] {
        accumulate_flow_forward_kernel<scalar_t,scalar_t><<<nblocks, nthreads>>>(
         fflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
         bflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
         pfflow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
         pbflow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
         stride0,locs_per_thread);
        }));
  }

}

/*******************************************


             Backward Flow


*******************************************/

template <typename scalar_t>
__device__ __forceinline__ 
void assign_bilin2d(scalar_t dAdF0[2], scalar_t dAdF1[2],
                    scalar_t gv0, scalar_t gv1, scalar_t* prop, int H, int W,
     torch::TensorAccessor<scalar_t,3,torch::RestrictPtrTraits,int32_t> g_flow){

  // -- read --
  int prop_i[2];
  scalar_t gH,gW,w;
#pragma unroll
  for (int ix=0;ix<2;ix++){
#pragma unroll
    for (int jx=0;jx<2;jx++){

      // -- interpolation weights --
      prop_i[0] = __float2int_rd(prop[1]+ix);
      gH = max(0.,1-fabs(prop_i[0]-prop[1]));
      prop_i[1] = __float2int_rd(prop[2]+jx);
      gW = max(0.,1-fabs(prop_i[1]-prop[2]));
      w = gH*gW;

      // -- bounds --
      prop_i[0] = bounds(prop_i[0],H);
      prop_i[1] = bounds(prop_i[1],W);

      // -- write --
      atomicAdd(&(g_flow[0][prop_i[0]][prop_i[1]]),w*(dAdF0[0]*gv0+ dAdF0[1]*gv1));
      atomicAdd(&(g_flow[1][prop_i[0]][prop_i[1]]),w*(dAdF1[0]*gv0+ dAdF1[1]*gv1));
      // atomicAdd(&(g_flow[0][prop_i[0]][prop_i[1]]),w*w0[0][1]*gv1);
      // atomicAdd(&(g_flow[1][prop_i[0]][prop_i[1]]),w*w1[0][0]*gv1);
      // atomicAdd(&(g_flow[1][prop_i[0]][prop_i[1]]),w*w1[0][1]*gv0);
      // atomicAdd(&(g_flow[0][prop_i[0]][prop_i[1]]),w*w0[ix][jx]*gv0);
      // atomicAdd(&(g_flow[1][prop_i[0]][prop_i[1]]),w*w1[ix][jx]*gv1);
      // atomicAdd(&(g_flow[0][prop_i[0]][prop_i[1]]),w*gv0);
      // atomicAdd(&(g_flow[1][prop_i[0]][prop_i[1]]),w*gv1);

    }
  }

}

template <typename scalar_t>
__device__ __forceinline__ 
void set_to_const(scalar_t w0[][2], scalar_t w1[][2], scalar_t C){
#pragma unroll
    for (int _ix = 0; _ix < 2; _ix++){
#pragma unroll
      for (int _jx = 0; _jx < 2; _jx++){
        w0[_ix][_jx] = C;
        w1[_ix][_jx] = C;
      }
    }
}

template <typename scalar_t>
__device__ __forceinline__ 
void update_weights(scalar_t dAdF0[2], scalar_t dAdF1[2],
                    scalar_t* prop, int H, int W, int tx,
    const torch::TensorAccessor<scalar_t,3,torch::RestrictPtrTraits,int32_t> flow,
    torch::TensorAccessor<scalar_t,3,torch::RestrictPtrTraits,int32_t> dev){

  // -- read --
  scalar_t dFlow[2][2];
  int prop_i[2];
  dFlow[0][0] = 0;
  dFlow[0][1] = 0;
  dFlow[1][0] = 0;
  dFlow[1][1] = 0;
  bool any_zero[2];
  any_zero[0] = false;
  any_zero[1] = false;

  // -- check bounds --
  int sH = check_interval(prop[1],0,H) ? 1 : -1;
  int sW = check_interval(prop[2],0,W) ? 1 : -1;

  dev[0][0][2] = prop[1];
  dev[0][0][3] = prop[2];
  dev[0][0][4] = sH;
  dev[0][0][5] = sW;

  // -- init wrap --
  prop[1] = bounds(prop[1],H);
  prop[2] = bounds(prop[2],W);

  scalar_t gH,gW,vW,vH;
#pragma unroll
  for (int ix=0;ix<2;ix++){
#pragma unroll
    for (int jx=0;jx<2;jx++){

      // -- interpolation weights --
      // prop_i[0] = __float2int_rz(prop[1]+ix);
      prop_i[0] = __float2int_rz(ix ==0 ? floorf(prop[1]) : ceilf(prop[1]));
      gH = max(0.,1-fabs(prop_i[0]-prop[1]));
      // prop_i[1] = __float2int_rz(prop[2]+jx);
      prop_i[1] = __float2int_rz(jx ==0 ? floorf(prop[2]) : ceilf(prop[2]));
      gW = max(0.,1-fabs(prop_i[1]-prop[2]));

      // -- compute direction --
      bool left0 = (prop_i[0]-prop[1]) < 0;
      bool right0 = (prop_i[0]-prop[1]) > 0;
      bool left1 = (prop_i[1]-prop[2]) < 0;
      bool right1 = (prop_i[1]-prop[2]) > 0;

      // zero out edge
      any_zero[0] = (not(left0) && not(right0)) or any_zero[0];
      any_zero[1] = (not(left1) && not(right1)) or any_zero[1];
      // left1 = jx == 0;
      // right1 = jx == 1;

      // -- bounds --
      prop_i[0] = bounds(prop_i[0],H);
      prop_i[1] = bounds(prop_i[1],W);
      // assert (prop_i[0]>=0);
      // assert (prop_i[1]>=0);

      // -- read --
      vW = flow[0][prop_i[0]][prop_i[1]];
      vH = flow[1][prop_i[0]][prop_i[1]];
      // if ((prop_i[0] < 2) && (prop_i[1] < 2)){
      //   vH = static_cast<scalar_t>(1/2.);
      // }else{
      //   vH = static_cast<scalar_t>(1/10.);
      // }
      // vH = static_cast<scalar_t>(1/10.);

      // -- write --
      dev[ix][jx][0] = prop_i[0];
      dev[ix][jx][1] = prop_i[1];
      // dev[ix][jx][4] = vH;
      // dev[ix][jx][5] = vW;

      // -- update --
      dFlow[0][0] += left1 ? -gH*vW : (right1 ? gH*vW : 0); // dF[0]/dF[0]; A(0)
      dFlow[0][1] += left0 ? -gW*vW : (right0 ? gW*vW : 0); // dF[0]/dF[0]; A(1)

      dFlow[1][0] += left1 ? -gH*vH : (right1 ? gH*vH : 0); // dF[1]/dF[1]; A(0)
      dFlow[1][1] += left0 ? -gW*vH : (right0 ? gW*vH : 0); // dF[1]/dF[1]; A(1)

    }
  }
  
  if(any_zero[0]){
    dFlow[0][1] = 0;
    dFlow[1][1] = 0;
  }
  if(any_zero[1]){
    dFlow[0][0] = 0;
    dFlow[1][0] = 0;
  }

  // -- reset or accumulate --
  if (tx == 0){
    // dFlow[0] = 1;
    // dFlow[1] = 1;
    dAdF0[0] = dFlow[0][0]*dAdF0[0] + dFlow[0][1]*dAdF0[1];
    dAdF0[1] = dFlow[1][0]*dAdF0[0] + dFlow[1][1]*dAdF0[1];
    dAdF1[0] = dFlow[0][0]*dAdF1[0] + dFlow[0][1]*dAdF1[1];
    dAdF1[1] = dFlow[1][0]*dAdF1[0] + dFlow[1][1]*dAdF1[1];
  }else{
    // dFlow[0] = 0;
    // dFlow[1] = 0;
    // int tmp0 = dAdF0[0][0];
    // int tmp1 = dAdF1[0][0];
    // dAdF0[0][0] += dFlow[0][0]*dAdF0[0][0] + dFlow[0][1]*dAdF0[0][0];
    // dAdF1[0][0] += dFlow[1][0]*dAdF1[0][0] + dFlow[1][1]*dAdF1[0][0];
    
    // -- assign --
    scalar_t _dAdF0[2];
    scalar_t _dAdF1[2];
    _dAdF0[0] = dAdF0[0];
    _dAdF0[1] = dAdF0[1];
    _dAdF1[0] = dAdF1[0];
    _dAdF1[1] = dAdF1[1];

    // -- update --
    dAdF0[0] += dFlow[0][0]*_dAdF0[0] + dFlow[0][1]*_dAdF0[1];
    dAdF0[1] += dFlow[1][0]*_dAdF0[0] + dFlow[1][1]*_dAdF0[1];
    dAdF1[0] += dFlow[0][0]*_dAdF1[0] + dFlow[0][1]*_dAdF1[1];
    dAdF1[1] += dFlow[1][0]*_dAdF1[0] + dFlow[1][1]*_dAdF1[1];

    // dAdF1[1] += (tx == 1) ? dFlow[1][1] : dFlow[1][1] * dAdF1[1];//*dAdF1[0][0];
    // dAdF0[0][0] += dFlow[0]*dAdF0[0][0];
    // dAdF1[0][0] += dFlow[1]*dAdF1[0][0];
  }

  // -- update using bounds --
  // dAdF0[0] = sW*dAdF0[0];
  // dAdF0[1] = sW*dAdF0[1];
  // dAdF1[0] = sH*dAdF1[0];
  // dAdF1[1] = sH*dAdF1[1];


}



template <typename scalar_t>
__global__ void accumulate_flow_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,7,torch::RestrictPtrTraits> dev,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_fflow,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_bflow,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> grad_pfflow,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> grad_pbflow,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fflow,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> bflow,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> pfflow,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> pbflow,
    int stride0, int nW, int nHW, int TnHW, int locs_per_thread){

  // -- unpack --
  int qi;
  int ibatch = blockIdx.y;
  int raster_index = locs_per_thread*(threadIdx.x + blockDim.x * blockIdx.x);
  int dir = threadIdx.y;
  int t_flow = threadIdx.z;
  int T = fflow.size(1);
  int H = fflow.size(3);
  int W = fflow.size(4);
  int tmp;
  int ref[3];
  scalar_t refs[3];
  int prop_i[3];
  scalar_t prop[3];
  scalar_t prop_time;

  // -- fwd decl --
  scalar_t v0,v1,gv0,gv1;
  scalar_t dAdF0[2];
  scalar_t dAdF1[2];

  // -- get directional information --
  if (dir != 0){ return; }
  auto flow = dir == 0 ? fflow[ibatch] : bflow[ibatch];
  auto g_flow = dir == 0 ? grad_fflow[ibatch] : grad_bflow[ibatch];
  auto pflow = dir == 0 ? pfflow[ibatch] : pbflow[ibatch];
  auto g_pflow = dir == 0 ? grad_pfflow[ibatch] : grad_pbflow[ibatch];

  // -- get location --
  for (int loc = 0; loc < locs_per_thread; loc++){

    // -- get reference location --
    qi = raster_index + loc;
    if (qi >= TnHW){ break; } 
    get_pixel_loc(ref,qi,tmp,stride0,nW,nHW,H,W);

    // -- assignments --
    v0 = 0;
    v1 = 0;
    gv0 = 0;
    gv1 = 0;
    // set_to_const(dAdF0,dAdF1,static_cast<scalar_t>(0));

    int t_inc = dir == 0 ? 1 : -1;
    int t_start = t_flow;//ref[0] + t_flow*t_inc;

    // -- write location --
    refs[0] = __int2float_rn(ref[0]);
    int dt = t_flow-ref[0];
    // if (dt >= 2){ return; }
    // if (dt > 0){ return; }

    if ((dt < 0) && (dir == 0)){ return; }
    // if ((refs[0] > t_flow) && (dir == 0)){ return; }
    // if ((refs[0] < t_flow) && (dir == 1)){ return; }
    if (dir == 1){ return; }
    if (dt == 0){
      refs[1] = __int2float_rn(ref[1]);
      refs[2] = __int2float_rn(ref[2]);
    }else{
      refs[1] = ref[1] + pflow[ref[0]][dt-1][1][ref[1]][ref[2]];
      refs[2] = ref[2] + pflow[ref[0]][dt-1][0][ref[1]][ref[2]];
    }

    int time = 0;
    // int t_end = dir == 0 ? (T-1)-ref[0]-t_flow : ref[0]-t_flow;
    int t_end = dir == 0 ? (T-1)-t_flow : 0;//ref[0]-t_flow;

    // -- iterate across accumulated flows --
    for(int tx=0; tx < t_end; tx++){

      // -- read gradient --
      gv0 = g_pflow[ref[0]][dt+tx][0][ref[1]][ref[2]];
      gv1 = g_pflow[ref[0]][dt+tx][1][ref[1]][ref[2]];
      // gv0 = g_pflow[ref[0]][tx][0][ref[1]][ref[2]];
      // gv1 = g_pflow[ref[0]][tx][1][ref[1]][ref[2]];

      // -- update weights --
      // update_weights(dAdF0,dAdF1,prop,H,W,flow[ref[0]+tx+1]); // for next one
      // set_to_const(dAdF0,dAdF1,static_cast<scalar_t>(1));
      if (tx==0){
        // set_to_const(dAdF0,dAdF1,static_cast<scalar_t>(1));
        dAdF0[0] = 1;
        dAdF0[1] = 0;
        dAdF1[0] = 0;
        dAdF1[1] = 1;
        // update_weights(dAdF0,dAdF1,prop,H,W,flow[tx+1]); // for next one
      }else{
        // set_to_const(dAdF0,dAdF1,static_cast<scalar_t>(0));
        update_weights(dAdF0,dAdF1,prop,H,W,tx,flow[t_flow+tx],
                       dev[ibatch][qi][t_flow][tx]); // for next one
      }
      // set_to_const(dAdF0,dAdF1,static_cast<scalar_t>(1));

      // -- assign to each of the 4 interpolated flow values --
      assign_bilin2d(dAdF0,dAdF1,gv0,gv1,refs,H,W,g_flow[t_flow]);

      // -- update proposed location --
      prop[1] = __int2float_rn(ref[1]) + pflow[ref[0]][dt+tx][1][ref[1]][ref[2]];
      prop[2] = __int2float_rn(ref[2]) + pflow[ref[0]][dt+tx][0][ref[1]][ref[2]];
      // prop[1] = __int2float_rn(ref[1]);
      // prop[2] = __int2float_rn(ref[2]);
      // prop[1] = __int2float_rn(ref[1]) + 0.1;
      // prop[2] = __int2float_rn(ref[2]) + 0.1;


      // if (tx >= 0){ break; }
      // prop[1] = ref[1] + flow[0][1][ref[1]][ref[2]];
      // prop[2] = ref[2] + flow[0][0][ref[1]][ref[2]];
      // prop[1] = ref[1];
      // prop[2] = ref[2];

    }

  }
}


void accumulate_flow_backward_cuda(
     torch::Tensor dev,
     torch::Tensor grad_fflow, torch::Tensor grad_bflow,
     const torch::Tensor grad_pfflow, const torch::Tensor grad_pbflow,
     const torch::Tensor fflow, const torch::Tensor bflow,
     const torch::Tensor pfflow, const torch::Tensor pbflow, int stride0){
  
  // -- unpack --
  int B = fflow.size(0);
  int T = fflow.size(1);
  int H = fflow.size(3);
  int W = fflow.size(4);

  // -- num 2 run --
  int nH = (H-1)/stride0+1;
  int nW = (W-1)/stride0+1;
  int nHW = nH*nW;
  int nRun = T*nH*nW;

  // -- kernel params --
  int locs_per_thread = 1;
  int _nthreads = 256/T;
  dim3 nthreads(_nthreads,2,T-1); // forward and backward
  int _nblocks = (nRun-1)/(_nthreads*locs_per_thread)+1;
  dim3 nblocks(_nblocks,B);
  // fprintf(stdout,"nblocks,nthreads: %d,%d\n",_nblocks,_nthreads);
  // fprintf(stdout,"stride0: %d\n",stride0);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(fflow.type(), "accumulate_flow_backward_kernel", ([&] {
     accumulate_flow_backward_kernel<scalar_t><<<nblocks, nthreads>>>(
       dev.packed_accessor32<scalar_t,7,torch::RestrictPtrTraits>(),
       grad_fflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
       grad_bflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
       grad_pfflow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
       grad_pbflow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
       fflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
       bflow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
       pfflow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
       pbflow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
       stride0,nW,nHW,nRun,locs_per_thread);
      }));
}

