#include "hip/hip_runtime.h"

#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cuda/std/type_traits>
#include <cstdio>
// #include <cuda/std/type_traits>
// #include <cstdio>

inline
at::ScalarType get_type(const torch::Tensor my_tensor){
  const auto& the_type = my_tensor.type();
  at::ScalarType _st = ::detail::scalar_type(the_type);
  return _st;
}

template< class T, class U >
inline constexpr bool is_same_v = ::cuda::std::is_same<T, U>::value;

__inline__ __device__ int bounds(int val, int lim ){
  int vval = val;
  if (val < 0){
    vval = -val;
  }else if (val >= lim){
    vval = 2*(lim-1)-val;
  }
  return vval;
}

__device__ __forceinline__
bool check_interval(int val, int lower, int upper){
  return (val >= lower) && (val < upper);
}

__inline__ __device__ int bounds2(int val, int lb, int ub){
  int vval = val;
  if ((val < lb) && (lb > 0)){
    vval = 2*lb - val;
  }else if (val < 0){
    vval = -val;
  }else if (val >= ub){
    vval = 2*(ub-1)-val;
  }
  return vval;
}



template<typename itype=int>
__device__ __forceinline__ 
void get_pixel_loc(itype* pix,  int qindex, int tmp, int stride0,
                   int nW0, int nHW0, int H, int W){
  int nH_index;
  if (is_same_v<itype,int>){
    tmp = qindex;
    pix[0] = tmp / nHW0;
    tmp = (tmp - pix[0]*nHW0); 
    nH_index = tmp / nW0;
    pix[1] = (nH_index*stride0) % H;
    tmp = tmp - nH_index*nW0;
    pix[2] = ((tmp % nW0) * stride0) % W;
  }else{
    tmp = qindex;
    pix[0] = round(tmp/nHW0);
    tmp = (tmp - pix[0]*nHW0); 
    nH_index = tmp / nW0;
    pix[1] = round((nH_index*stride0) % H);
    tmp = tmp - nH_index*nW0;
    pix[2] = round(((tmp % nW0) * stride0) % W);
  }
}

