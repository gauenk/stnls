#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

/****************************

       Helper Funcs

****************************/

__inline__ __device__ int bounds(int val, int lim ){
  int vval = val;
  if (val < 0){
    vval = -val;
  }else if (val >= lim){
    vval = 2*(lim-1) - val;
  }
  return vval;
}

__inline__ int cpu_bounds(int val, int lim ){
  int vval = val;
  if (val < 0){
    vval = -val;
  }else if (val >= lim){
    vval = 2*(lim-1) - val;
  }
  return vval;
}

/****************************

       Forward Pass

****************************/

template <typename scalar_t>
__global__ void wpsum_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> vid,
    torch::PackedTensorAccessor32<scalar_t,7,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<int,4,torch::RestrictPtrTraits> inds,
    int h_off, int w_off, int dilation, int adj, bool reflect_bounds, int qpt, int cpt){

    // -- shapes --
    int nbatch = vid.size(0);
    int nframes = vid.size(1);
    int colors = vid.size(2);
    int height = vid.size(3);
    int width = vid.size(4);
    int nq = patches.size(1);
    int k = inds.size(2);
    int pt = patches.size(3);
    int ps = patches.size(5);
    int psHalf = (int)ps/2;
    int center_ti,center_hi,center_wi;

    // -- cuda threads --
    int pi = threadIdx.y;
    int pj = threadIdx.z;

    // -- batching --
    int query_start = blockIdx.x*qpt;
    int c_start = threadIdx.x*cpt;
    int ibatch = blockIdx.y;

    // inits
    int qi,ki,ci;
    int ti,hi,wi;
    bool valid_hw,valid_t,valid;
    scalar_t pix,dist;

    // -- range --
    for(int _qi = 0; _qi < qpt; _qi++){

      // -- query index --
      qi = query_start + _qi;
      if (qi >= nq){ continue; }

      for(int ki = 0; ki < k; ki++){

        // -- reference center --
        center_ti = inds[ibatch][qi][ki][0];
        center_hi = inds[ibatch][qi][ki][1];
        center_wi = inds[ibatch][qi][ki][2];
        dist = dists[ibatch][qi][ki];

        // -- reference patch location --
        if (reflect_bounds){
          hi = bounds((center_hi-h_off)+dilation*(pi - psHalf + adj),height);
          wi = bounds((center_wi-w_off)+dilation*(pj - psHalf + adj),width);
        }else{
          hi = (center_hi-h_off)+dilation*(pi - psHalf + adj);
          wi = (center_wi-w_off)+dilation*(pj - psHalf + adj);
        }

        // -- spatially valid --
        valid_hw = (hi >= 0) && (hi < height);
        valid_hw = valid_hw && (wi >= 0) && (wi < width);

        // -- iterate over loop --
        for(int pk = 0; pk < pt; pk++){

          // -- check valid --
          ti = bounds(center_ti + pk,nframes);
          valid_t = (ti >= 0) && (ti < nframes);
          valid = valid_hw && valid_t;

          // -- colors --
          for(int _ci = 0; _ci < cpt; _ci++){

            // -- color index --
            ci = c_start + _ci;

            // -- fill without warp divergence --
            if (valid && (ci < colors)){
              pix = dist*vid[ibatch][ti][ci][hi][wi];
              patches[ibatch][qi][0][pk][ci][pi][pj] += pix;
            }

          }
        }
      }
    }
}

void cuda_wpsum_forward(
    torch::Tensor vid, torch::Tensor patches,
    torch::Tensor dists, torch::Tensor inds,
    int h_off, int w_off, int dilation, int adj, bool reflect_bounds){

  // -- kernel blocks --
  int nbatch = inds.size(0);
  int nqueries = inds.size(1);
  int qpt = 3;
  int nblocks_queries = (nqueries-1)/qpt+1;
  dim3 nblocks(nblocks_queries,nbatch);

  // -- kernel threads --
  int k = inds.size(2);
  int colors = vid.size(2);
  int ps = patches.size(6);
  int MAX_THREADS = 1024;
  int dim = ps*ps;
  int cpb = MAX_THREADS/dim; // num of colors per block
  int cpt = ((colors - 1)/cpb) + 1; // num of colors per thread
  dim3 nthreads(cpb,ps,ps);
  // printf("colors: %d, cpt: %d, cpb: %d, ps: %d, nblocks: %d, rbounds: %d\n",
  //        colors,cpt,cpb,ps,nblocks,(int)reflect_bounds);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(vid.type(), "wpsum_forward_kernel", ([&] {
    wpsum_forward_kernel<scalar_t><<<nblocks, nthreads>>>(
        vid.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<scalar_t,7,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,4,torch::RestrictPtrTraits>(),
        h_off, w_off, dilation, adj, reflect_bounds, qpt, cpt);
    }));
}

/********************************

     Backward Pass (for Vid)

********************************/


template <typename scalar_t>
__global__ void wpsum_backward_vid_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> vid_grad,
    const torch::PackedTensorAccessor32<scalar_t,7,torch::RestrictPtrTraits> patches_grad,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<int,4,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> rand_nums,
    int h_off, int w_off, int dilation, int adj, bool reflect_bounds, int qpt, int cpt){

  // shape
  int nbatch = patches_grad.size(0);
  int nq =    patches_grad.size(1);
  int k =     inds.size(2);
  int pt =    patches_grad.size(3);
  int colors = patches_grad.size(4);
  int ps =    patches_grad.size(5);
  int qi,ti,hi,wi;
  float weight,pix;
  int height = vid_grad.size(3);
  int width = vid_grad.size(4);
  int psHalf = ps/2;
  bool valid_h,valid_w,valid;
  int center_ti,center_hi,center_wi;

  // color indices
  int c0_start = threadIdx.y*cpt;
  int c0_end = min(c0_start + cpt,colors);
  int c0 = 0;
  int c0_offset = 0;
  int c0_dist = c0_end - c0_start;

  // block indices
  int thread_x = threadIdx.x;
  int block_x = blockIdx.x;
  int q_start = qpt*( thread_x + block_x * blockDim.x);
  int ibatch = blockIdx.y;
  
  for (int _qi = 0; _qi < qpt; _qi++){
    qi = q_start + _qi;
    if (qi < nq){
      c0_offset = __float2int_rd(c0_dist * rand_nums[qi][0][0]);
      // iterate
      for (int ki = 0; ki < k; ki++){
        c0_offset = (c0_offset + 1) % c0_dist;
        center_ti = inds[ibatch][qi][ki][0];
        center_hi = inds[ibatch][qi][ki][1];
        center_wi = inds[ibatch][qi][ki][2];
        for (int pk = 0; pk < pt; pk++){
          for (int pi = 0; pi < ps; pi++){
            for (int pj = 0; pj < ps; pj++){
              ti = center_ti + pk;
              hi = (center_hi-h_off) + dilation*(pi - psHalf + adj);
              wi = (center_wi-w_off) + dilation*(pj - psHalf + adj);
              hi = reflect_bounds ? bounds(hi,height) : hi;
              wi = reflect_bounds ? bounds(wi,width) : wi;
              valid_h = (hi >= 0) && (hi < height);
              valid_w = (wi >= 0) && (wi < width);
              valid = valid_h && valid_w;
              weight = dists[ibatch][qi][ki];
              for (int _c0 = c0_start; _c0 < c0_end; _c0++){
                c0 = (_c0 + c0_offset) % c0_dist + c0_start;
                pix = weight * patches_grad[ibatch][qi][0][pk][c0][pi][pj];
                if (valid){
                  vid_grad[ibatch][ti][c0][hi][wi] += pix;
                }
              }
            }
          }
        }
      }
    }
  }
}

void cuda_wpsum_backward_vid(
    torch::Tensor vid_grad, torch::Tensor patches_grad, 
    torch::Tensor dists, torch::Tensor inds,
    int h_off, int w_off, int dilation, int adj, bool reflect_bounds, bool exact){

  // unpack params
  int nbatch = inds.size(0);
  int numQueries = inds.size(1);
  int k = dists.size(2);
  int pt = patches_grad.size(3);
  int colors = patches_grad.size(4);
  int ps = patches_grad.size(5);
  assert(pt == 1);

  // num of threads
  int max_nthreads = 1024;
  int color_threads = 1;
  int block_threads = max_nthreads/color_threads;
  int cpt = (colors-1)/color_threads+1;
  block_threads = exact ? 1 : block_threads;
  color_threads = exact ? colors : color_threads;
  dim3 nthreads(block_threads,color_threads);

  // num of blocks
  int max_nblocks = 32;
  int num_per_block = 16;
  int total_per_block = block_threads * num_per_block;
  int nblocks_queries = ((numQueries - 1) / total_per_block) + 1;
  nblocks_queries = min(nblocks_queries,max_nblocks);
  int total_pb = (numQueries - 1) / nblocks_queries + 1;
  int bpb = (total_pb-1) / block_threads + 1;

  // exact gradient
  if (exact){
    cpt = 1;
    nblocks_queries = 1;
    block_threads = 1;
    bpb = numQueries;
  }
  dim3 nblocks(nblocks_queries,nbatch);

  // -- viz --
  // fprintf(stdout,"nblocks,block_threads,color_threads: %d,%d,%d\n",nblocks,block_threads,color_threads);
  // fprintf(stdout,"bpb,cpt: %d,%d\n",bpb,cpt);

  // -- allocate random memory --
  auto cu_index = vid_grad.device().index();
  auto options = torch::TensorOptions().device(torch::kCUDA,
                                               cu_index).dtype(torch::kFloat32);
  torch::Tensor rand_nums = torch::rand({numQueries,1,1},options);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(vid_grad.type(), "wpsum_backward_vid_kernel", ([&] {
    wpsum_backward_vid_kernel<scalar_t><<<nblocks, nthreads>>>(
        vid_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        patches_grad.packed_accessor32<scalar_t,7,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,4,torch::RestrictPtrTraits>(),
        rand_nums.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        h_off,w_off,dilation, adj, reflect_bounds, bpb, cpt);
  }));
    
}


/********************************

    Backward Pass (for Dists)

********************************/


template <typename scalar_t>
__global__ void wpsum_backward_dists_kernel(
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> dists_grad,
    const torch::PackedTensorAccessor32<scalar_t,7,torch::RestrictPtrTraits> patches_grad,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> vid,
    const torch::PackedTensorAccessor32<int,4,torch::RestrictPtrTraits> inds,
    int h_off, int w_off, int dilation, int adj, bool reflect_bounds){

  // -- shapes --
  int nbatch = dists_grad.size(0);
  int nq = dists_grad.size(1);
  int k = dists_grad.size(2);
  int pt =    patches_grad.size(3);
  int colors = patches_grad.size(4);
  int ps =    patches_grad.size(5);
  int height = vid.size(3);
  int width = vid.size(4);
  int psHalf = ps/2;

  // -- init registers --
  int ti,hi,wi;
  float pix_n,pix_m;
  bool valid_h,valid_w,valid;

  // -- location to fill --
  int qi = blockIdx.x*blockDim.x+threadIdx.x;
  int ki = blockIdx.y*blockDim.y+threadIdx.y;
  int ibatch = blockIdx.z;

  if ((qi < nq) && (ki < k)) { // -- if valid --
    int center_ti = inds[ibatch][qi][ki][0];
    int center_hi = inds[ibatch][qi][ki][1];
    int center_wi = inds[ibatch][qi][ki][2];
    for (int pk = 0; pk < pt; pk++){
      ti = center_ti + pk;
      for (int pi = 0; pi < ps; pi++){
        hi = (center_hi-h_off) + dilation*(pi - psHalf + adj);
        hi = reflect_bounds ? bounds(hi,height) : hi;
        valid_h = (hi >= 0) && (hi < height);
        for (int pj = 0; pj < ps; pj++){
          wi = (center_wi-w_off) + dilation*(pj - psHalf + adj);
          wi = reflect_bounds ? bounds(wi,width) : wi;
          valid_w = (wi >= 0) && (wi < width);
          valid = valid_h && valid_w;
          for (int c0 = 0; c0 < colors; c0++){
              pix_n = patches_grad[ibatch][qi][0][pk][c0][pi][pj];
              pix_m = valid ? vid[ibatch][ti][c0][hi][wi] : 0;
              dists_grad[ibatch][qi][ki] += valid ? pix_n * pix_m : 0.;
          }
        }
      }
    }
  }

}

void cuda_wpsum_backward_dists(
    torch::Tensor dists_grad, torch::Tensor patches_grad,
    torch::Tensor vid, torch::Tensor inds,
    int h_off, int w_off, int dilation, int adj, bool reflect_bounds, bool exact){

  // const int NQ,NK = 4,4;
  int nbatch = dists_grad.size(0);
  int nq = dists_grad.size(1);
  int k = dists_grad.size(2);
  dim3 threadsPerBlock(32,32);
  dim3 blocksPerGrid(1, 1);
  blocksPerGrid.x = ceil(double(nq)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(k)/double(threadsPerBlock.y));

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(vid.type(), "wpsum_backward_dists_kernel", ([&] {
    wpsum_backward_dists_kernel<scalar_t><<<blocksPerGrid, threadsPerBlock>>>(
        dists_grad.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        patches_grad.packed_accessor32<scalar_t,7,torch::RestrictPtrTraits>(),
        vid.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,4,torch::RestrictPtrTraits>(),
        h_off,w_off,dilation, adj, reflect_bounds);
  }));
    
}



/*********************************************

                 Misc

*********************************************/

template <typename scalar_t>
void wpsum_backward_exact(
    torch::Tensor _vid_grad,
    torch::Tensor _patches_grad,
    torch::Tensor _dists,
    torch::Tensor _inds,
    int h_off, int w_off,
    int dilation, int adj, bool reflect_bounds){

  // get accessors 
  auto vid_grad = _vid_grad.accessor<scalar_t,4>();
  auto patches_grad = _patches_grad.accessor<scalar_t,6>();
  auto dists = _dists.accessor<scalar_t,2>();
  auto inds = _inds.accessor<int,3>();
  
  // shape
  int nq =    patches_grad.size(0);
  int k =     inds.size(1);
  int pt =    patches_grad.size(2);
  int colors = patches_grad.size(3);
  int ps =    patches_grad.size(4);
  int nframes = vid_grad.size(0);
  int height = vid_grad.size(2);
  int width = vid_grad.size(3);
  int psHalf = ps/2;
  bool valid_h,valid_w,valid;

  int ti,hi,wi;
  int center_ti,center_hi,center_wi;
  float weight,pix,dist;

  for (int qi = 0; qi < nq; qi++){
    for (int ki = 0; ki < k; ki++){


      // -- center location --
      center_ti = inds[qi][ki][0];
      center_hi = inds[qi][ki][1];
      center_wi = inds[qi][ki][2];
      dist = dists[qi][ki];

      for (int pk = 0; pk < pt; pk++){
        for (int pi = 0; pi < ps; pi++){
          for (int pj = 0; pj < ps; pj++){

            // -- pix location --
            ti = cpu_bounds(center_ti + pk,nframes);
            hi = (center_hi-h_off) + dilation*(pi - psHalf + adj);
            wi = (center_wi-w_off) + dilation*(pj - psHalf + adj);
            hi = reflect_bounds ? cpu_bounds(hi,height) : hi;
            wi = reflect_bounds ? cpu_bounds(wi,width) : wi;

            // -- check valid --
            valid_h = (hi >= 0) && (hi < height);
            valid_w = (wi >= 0) && (wi < width);
            valid = valid_h && valid_w;

            for (int ci = 0; ci < colors; ci++){
              pix = dist * patches_grad[qi][0][pk][ci][pi][pj];
              if (valid){
                vid_grad[ti][ci][hi][wi] += pix;
              }
            }
          }
        }
      }

    }
  }

}


