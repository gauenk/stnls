#include "hip/hip_runtime.h"



// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

/****************************

       Helper Funcs

****************************/

#define CUDA_KERNEL_LOOP_TYPE(i, n, index_type)                         \
  int64_t _i_n_d_e_x = blockIdx.x * blockDim.x + threadIdx.x;           \
  for (index_type i=_i_n_d_e_x; _i_n_d_e_x < (n); _i_n_d_e_x+=blockDim.x * gridDim.x, i=_i_n_d_e_x)

#define CUDA_KERNEL_LOOP(i, n) CUDA_KERNEL_LOOP_TYPE(i, n, int)

__inline__ __device__ int bounds(int val, int lb, int ub ){
  int vval = val;
  if (val < lb){
    vval = 2*lb - val;
  }else if (val >= ub){
    vval = 2*(ub-1) - val;
  }
  return vval;
}

/****************************

       Forward Pass

****************************/

template <typename scalar_t>
__global__ void dnls_iunfold_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> patches,
    int top, int left, int btm, int right,
    int start, int stride, int dilation, int adj,
    bool only_full, bool use_reflect, int qpt, int kpt) {

    // -- shapes --
    int nframes = vid.size(0);
    int colors = vid.size(1);
    int height = vid.size(2);
    int width = vid.size(3);
    int nq = patches.size(0);
    int k = patches.size(1);
    int pt = patches.size(2);
    int ps = patches.size(4);
    int psHalf = (int)ps/2;
    int height_width = height*width;
    int fill_pad = dilation * psHalf;
    int dil = dilation;

    // -- cuda threads --
    int pi = threadIdx.y;
    int pj = threadIdx.z;

    // -- batching --
    int query_start_block = blockIdx.x*qpt;
    int k_start = threadIdx.x*kpt;

    // -- unpack --
    int sq_h = btm - top;
    int sq_w = right - left;
    int sq_hw = sq_h * sq_w;

    // -- strided size --
    int n_h = int((sq_h-1) / stride) + 1;
    int n_w = int((sq_w-1) / stride) + 1;
    if (only_full){
      n_h = (sq_h - (ps-1)*dil - 1)/stride + 1;
      n_w = (sq_w - (ps-1)*dil - 1)/stride + 1;
    }
    int n_hw = n_h*n_w;

    // inits
    int qIndex,_qIndex;
    int qi,ki,ti,hi,wi,qi_mod;
    int vi_h,vi_w,vi_t;
    bool valid_hw,valid_t,valid;
    scalar_t pix;
    int stride2 = stride * stride;

    // -- range --
    for(int _qi = 0; _qi < qpt; _qi++){

      // -- query index --
      qi = _qi + query_start_block;
      if (qi >= nq){ continue; }

      for(int _ki = 0; _ki < kpt; _ki++){

        // -- k index --
        ki = k_start + _ki;
        if (ki >= k){ continue; }

        // -- fill --
        // qIndex = qi*stride;
        // wi = qIndex % width;
        // hi = (qIndex/width) % height;
        // ti = (qIndex/heigh_width) % nframes;

        // -- ind with stride --
        // qIndex = stride2*(qi + start);
        // ti = (qIndex/height_width) % nframes;
        // _qIndex = qIndex % height_width;
        // hi = (stride)*(_qIndex / (stride*width)) % height;
        // wi = (_qIndex/stride) % width;

        // -- new inds --
        qIndex = qi + start;
        ti = qIndex / (n_hw);
        qi_mod = qIndex % (n_hw);
        hi = (qi_mod / n_w) * stride + top;
        wi = (qi_mod % n_w) * stride + left;

        // -- valid ind --
        valid_hw = (hi >= -fill_pad) && (hi < (height+fill_pad));
        valid_hw = valid_hw && (wi >= -fill_pad) && (wi < (width+fill_pad));
        // valid_hw = (hi >= (top-fill_pad)) && (hi < (btm+fill_pad));
        // valid_hw = valid_hw && (wi >= (left-fill_pad)) && (wi < (right+fill_pad));
        valid_hw = valid_hw && (ti   >= 0) && (ti < nframes);

        // -- fill across cuda threads --
        // vi_h = hi+dilation*(pi - psHalf);
        // vi_w = wi+dilation*(pj - psHalf);
        if (use_reflect){
          vi_h = bounds(hi+dilation*(pi - psHalf + adj),0.,height);
          vi_w = bounds(wi+dilation*(pj - psHalf + adj),0.,width);
        }else{
          vi_h = hi+dilation*(pi - psHalf + adj);
          vi_w = wi+dilation*(pj - psHalf + adj);
        }
        // vi_h = bounds(hi+dilation*(pi - psHalf),top,btm);
        // vi_w = bounds(wi+dilation*(pj - psHalf),left,right);

        // -- spatially valid --
        valid_hw = valid_hw && (vi_h >= 0) && (vi_h < height);
        valid_hw = valid_hw && (vi_w >= 0) && (vi_w < width);
        // valid_hw = valid_hw && (vi_h >= top) && (vi_h < btm);
        // valid_hw = valid_hw && (vi_w >= left) && (vi_w < right);


        // -- iterate over loop --
        for(int pk = 0; pk < pt; pk++){

          // -- check valid --
          vi_t = bounds(ti + pk,0,nframes);
          valid_t = (vi_t >= 0) && (vi_t < nframes);
          valid = valid_hw && valid_t;

          // -- colors --
          for(int ci = 0; ci < colors; ci++){
            if (valid){
              pix = vid[vi_t][ci][vi_h][vi_w];
            }else{
              pix = 0.;
            }
            patches[qi][ki][pk][ci][pi][pj] = pix;
          }
        }
      }
    }
}

void dnls_cuda_iunfold_forward(
    torch::Tensor vid, torch::Tensor patches,
    int top, int left, int btm, int right,
    int start, int stride, int dilation,
    int adj, bool only_full, bool use_reflect){

  // -- kernel blocks --
  int numQueries = patches.size(0);
  int k = 1;
  int qpt = 4;
  int nblocks = (numQueries-1)/qpt+1;
  int pt = patches.size(2);
  assert(pt == 1);

  // -- kernel threads --
  int ps = patches.size(5);
  int MAX_THREADS = 1024;
  int dim = ps*ps;
  int kpb = MAX_THREADS/dim; // num of "k" managed per block
  int kpt = ((k - 1)/kpb) + 1; // num of "k" per thread
  dim3 nthreads(kpb,ps,ps);
  // fprintf(stdout,"top,left,btm,right: %d,%d,%d,%d\n",top,left,btm,right);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(patches.type(), "dnls_iunfold_forward_kernel", ([&] {
    dnls_iunfold_forward_kernel<scalar_t><<<nblocks, nthreads>>>(
        vid.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        top, left, btm, right, start, stride, dilation,
        adj, only_full, use_reflect, qpt, kpt);
      }));
}


/****************************

       Backward Pass

****************************/

template <typename scalar_t>
__global__ void dnls_iunfold_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> patches,
    int top, int left, int btm, int right, int start, int stride, int dilation,
    int adj, bool only_full, bool use_reflect, int num_kernels) {

    // -- unpack --
    int nframes = vid.size(0);
    int colors = vid.size(1);
    int height = vid.size(2);
    int width = vid.size(3);
    int pt = patches.size(2);
    int ps = patches.size(5);
    int numQueries = patches.size(0);
    int psOffset = (ps-1)/2;
    int psHalf = ps/2;
    int hw = height*width;
    int fill_pad = psHalf * dilation;
    bool valid,valid_q,is_edge;
    int dil = dilation;

    // -- boundary --
    int right_a = right - (ps-1)*dil;
    int btm_a = btm - (ps-1)*dil;
    int right_bnd = (only_full) ? right_a : right;
    int btm_bnd = (only_full) ? btm_a : btm;

    // -- make square --
    int sq_h = btm - top;
    int sq_w = right - left;
    int sq_hw = sq_h * sq_w;

    // -- strided size --
    int n_h = int((sq_h-1) / stride) + 1;
    int n_w = int((sq_w-1) / stride) + 1;
    if (only_full){
      n_h = (sq_h - (ps-1)*dil - 1)/stride + 1;
      n_w = (sq_w - (ps-1)*dil - 1)/stride + 1;
    }
    int n_hw = n_h * n_w;

    CUDA_KERNEL_LOOP(_index, num_kernels) {

      int index = (_index);
      const int64_t t_im = (index / sq_hw);
      const int64_t i_mod = index % sq_hw;
      const int64_t w_im = i_mod % sq_w + left;
      const int64_t h_im = (i_mod / sq_w) % sq_h + top;
      // int index = (_index);
      // const int64_t w_im = index % width;
      // const int64_t h_im = (index / width) % height;
      // const int64_t t_im = (index / hw);

      // -- allow partial nhits if edge --
      // int padf = dilation*ps;
      // bool is_edge = (w_im < padf) || (w_im > (width-padf));
      // is_edge = is_edge || (h_im < padf) || (h_im > (height-padf));
        
      for(int ci = 0; ci < colors; ci++){
        // nhits = 0;
        // nhits_q = 0;
        scalar_t val = 0;
        for (int pk = 0; pk < pt; pk++){
          for (int pi = 0; pi < ps; pi++){
            for (int pj = 0; pj < ps; pj++){

              // -- offsets for ni --
              int _wi = w_im + dilation*(pi - psOffset - adj);
              int _hi = h_im + dilation*(pj - psOffset - adj);
              // int _wi = w_im + dilation*(pi - psHalf);
              // int _hi = h_im + dilation*(pj - psHalf);
              int ti = t_im + pk;

              // -- check bounds --
              valid = (_wi >= left) && (_wi < (right_bnd));
              valid = valid && (_hi >= top) && (_hi < (btm_bnd));

              // valid = (_wi >= (left-fill_pad)) && (_wi < (right+fill_pad));
              // valid = valid && (_hi >= (top-fill_pad)) && (_hi < (btm+fill_pad));

              // valid = (_wi >= -fill_pad) && (_wi < (width+fill_pad));
              // valid = valid && (_hi >= -fill_pad) && (_hi < (height+fill_pad));
              // int wi = bounds(_wi,left,right);
              // int hi = bounds(_hi,top,btm);
              int wi = _wi;
              int hi = _hi;
              // int wi = bounds(_wi,left,right);
              // int hi = bounds(_hi,top,btm);
              // int wi = bounds(_wi,0,width);
              // int hi = bounds(_hi,0,height);

              // -- compute ni --
              // int qi = ti * n_hw + ((hi/stride) * n_W)+ (wi/stride);
              // qi -= start;
              int qi = ti * n_hw;
              qi += (((hi-top)/stride) * n_w);
              qi += ((wi-left)/stride);
              qi -= start;

              // -- only if qi is aligned with center --
              valid = valid && ((hi-top) % stride == 0) && ((wi-left) % stride == 0);

              // -- patch indexing --
              int w_ip = ps-1-pi;
              int h_ip = ps-1-pj;

              // -- reflect to match --
              if (_wi > wi){
                w_ip = pi;
                valid = valid && (w_ip < psHalf);
              }
              else if(_wi < wi){
                w_ip = pi;
                valid = valid && (w_ip > psHalf);
              }

              if (_hi > hi){
                h_ip = pj;
                valid = valid && (h_ip < psHalf);
              }
              else if(_hi < hi){
                h_ip = pj;
                valid = valid && (h_ip > psHalf);
              }

              // -- accumulate --
              valid_q = valid && (qi >= 0) && (qi < numQueries);
              if (valid_q){
                val += patches[qi][0][0][ci][h_ip][w_ip];
                // nhits_q += 1;
              }
              // if(valid){
              //   nhits += 1;
              // }
            }
          } // for patch size
        } // for patch size
        // bool eq_hits = nhits == nhits_q;
        // bool hit_req = true;//((not is_edge) && (nhits == ndim)) || is_edge;
        // if (eq_hits){
        //   vid[t_im][ci][h_im][w_im] =  val;
        // }
        vid[t_im][ci][h_im][w_im] = val;
        // vid[t_im][ci][h_im][w_im] += val;
      } // for colors
    }
}

void dnls_cuda_iunfold_backward(
  torch::Tensor grad_vid,torch::Tensor patches,
  int top, int left, int btm, int right,
  int start, int stride, int dilation,
  int adj, bool only_full, bool use_reflect) {

  // -- kernel blocks --
  // int numQueries = patches.size(0);
  // int k = 1;
  int nframes = grad_vid.size(0);
  int height = grad_vid.size(2);
  int width = grad_vid.size(3);

  // make square
  int sq_h = btm - top;
  int sq_w = right - left;
  int sq_hw = sq_h * sq_w;


  // threads and blocks
  int nthreads = 512;
  // int num_kernels = patches.size(0);//nframes*height*width;
  // int num_kernels = nframes*height*width;
  int num_kernels = nframes*sq_hw;
  int nblocks = (num_kernels-1) / nthreads+1;

  // get starting pixel

  // -- kernel threads --
  // int ps = patches.size(5);
  // int MAX_THREADS = 1024;
  // int dim = ps*ps;
  // int kpb = MAX_THREADS/dim; // num of "k" managed per block
  // int kpt = ((k - 1)/kpb) + 1; // num of "k" per thread
  // dim3 nthreads(kpb,ps,ps);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(patches.type(), "dnls_iunfold_backward_kernel", ([&] {
    dnls_iunfold_backward_kernel<scalar_t>
      <<<nblocks, nthreads>>>(
        grad_vid.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        top, left, btm, right, start, stride, dilation,
        adj, only_full, use_reflect, num_kernels);
  }));

}
