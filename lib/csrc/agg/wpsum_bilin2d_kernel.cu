#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../shared_kernel.cu"

/****************************

       Forward Pass

****************************/

template <typename scalar_t>
__global__ void wpsum_bilin2d_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid,
    torch::PackedTensorAccessor32<int,2,torch::RestrictPtrTraits> counts,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds,
    int ps, int stride0, int pt, int dilation, bool reflect_bounds,
    int patch_offset, int q_per_thread){

    // -- shapes --
    int B = in_vid.size(0);
    int HD = in_vid.size(1);
    int T = in_vid.size(2);
    int F = in_vid.size(3);
    int H = in_vid.size(4);
    int W = in_vid.size(5);
    int Q = inds.size(2);
    int K = inds.size(3);

    // -- batching --
    int query_start = (threadIdx.x + blockDim.x*blockIdx.x)*q_per_thread;
    int ibatch = blockIdx.y;
    int ihead = blockIdx.z;

    // -- cuda threads --
    int pi = threadIdx.y;
    int pj = threadIdx.z;

    // -- pixel locations --
    int qi;
    bool valid;
    scalar_t pix,weight;
    int ref_ti,nl_ti;
    scalar_t nl[3];
    int ref[3],ref_p[3];
    int nW = (W-1)/stride0+1;
    int nHW = nW*((H-1)/stride0+1);


    // -- across queries --
    for(int _qi = 0; _qi < q_per_thread; _qi++){

      // -- query index --
      qi = query_start + _qi;
      if (qi >= Q){ continue; }
      get_pixel_loc<int>(ref,qi,stride0,nW,nHW,H,W);

      // -- reference pixel index --
      ref_p[0] = ref[0];
      ref_p[1] = ref[1]+dilation*(pi + patch_offset);
      ref_p[2] = ref[2]+dilation*(pj + patch_offset);

      // -- valid ref pixel only --
      check_bounds(valid, ref_p, T,  H, W);
      if (not valid){ continue; }

      // -- normalize --
      if ((ref[0]==0) and (ibatch==0) and (ihead==0)){
        atomicAdd(&counts[ref_p[1]][ref_p[2]],1);
      }

      for(int ki = 0; ki < K; ki++){

        // -- non-local index --
        #pragma unroll
        for (int _idx=0; _idx < 3; _idx++){
          nl[_idx] = ref[_idx] + inds[ibatch][ihead][qi][ki][_idx];
        }

        // -- always flow --
        nl[0] = bounds(nl[0],T);
        nl[1] = bounds(nl[1],H);
        nl[2] = bounds(nl[2],W);

        // -- non-local pixel index --
        nl[1] = nl[1]+dilation*(pi + patch_offset);
        nl[1] = reflect_bounds ? bounds(nl[1],H) : nl[1];
        nl[2] = nl[2]+dilation*(pj + patch_offset);
        nl[2] = reflect_bounds ? bounds(nl[2],W) : nl[2];

        // -- valid non-local patches only --
        check_bounds(valid, nl, T,  H, W);
        if (not valid){ continue; }

        // -- non-local weight --
        weight = dists[ibatch][ihead][qi][ki];

        // -- iterate over loop --
        for(int pk = 0; pk < pt; pk++){

          // -- time is always valid --
          ref_ti = ref_p[0] + pk;
          nl_ti = reflect_bounds ? bounds(nl[0]+pk,T) : (nl[0]+pk);
          valid = check_bound(nl_ti, T) and check_bound(ref_ti, T);
          if (not valid){ continue; }

          // -- channels --
          for(int iftr = 0; iftr < F; iftr++){

            // -- fill --
            bilin2d_interpolate(pix,nl[1],nl[2],H,W,
                                in_vid[ibatch][ihead][nl_ti][iftr]);
            atomicAdd(&out_vid[ibatch][ihead][ref_ti][iftr][ref_p[1]][ref_p[2]],
                      weight*pix);

          } // nfeatures-loop
        } // pt-loop
      } // k-loop
    } // query-loop
}

void wpsum_bilin2d_forward_cuda(
    torch::Tensor out_vid, torch::Tensor counts,
    const torch::Tensor in_vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int stride0, int pt, int dilation,
    bool reflect_bounds, int patch_offset){

  // -- unpack --
  int B = inds.size(0);
  int HD = inds.size(1);
  int Q = inds.size(2);
  int q_per_thread = 1;

  // -- kernel threads --
  int MAX_THREADS = 512;//1024
  int q_threads = MAX_THREADS/(ps*ps); // num of queries threads per block
  q_threads = min(Q,q_threads);
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  dim3 nthreads(q_threads,ps,ps);
  // fprintf(stdout,
  //         "ps,pt,stride0,reflect_bounds,dilation,patch_offset: %d,%d,%d,%d,%d,%d\n",
  //         ps,pt,stride0,reflect_bounds,dilation,patch_offset);

  // -- kernel blocks --
  dim3 nblocks(q_blocks,B,HD);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(in_vid.type(), "wpsum_bilin2d_forward_kernel", ([&] {
    wpsum_bilin2d_forward_kernel<scalar_t><<<nblocks, nthreads>>>(
        out_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        counts.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
        in_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        ps, stride0, pt, dilation, reflect_bounds, patch_offset,
        q_per_thread);
    }));
}



/************************************

  Backward Pass (for Vid & Dists)

*************************************/

template <typename scalar_t>
__global__ void wpsum_bilin2d_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds,
    int ps, int stride0, int pt, int dilation, bool reflect_bounds, int patch_offset,
    int q_per_thread, int k_per_thread){

  // -- shape --
  int B =  dists.size(0);
  int HD = dists.size(1);
  int Q =  dists.size(2);
  int K =  dists.size(3);
  int T = out_vid_grad.size(2);
  int F = out_vid_grad.size(3);
  int H = out_vid_grad.size(4);
  int W = out_vid_grad.size(5);

  // -- pixel indexing --
  int qi,ki;
  scalar_t nl[3];
  int ref[3],ref_p[3];
  int ref_ti,nl_ti;
  bool valid;
  scalar_t weight,grad,pix_m;

  // -- location to fill --
  int q_start = q_per_thread*(blockIdx.x*blockDim.x+threadIdx.x);
  int k_start = 0;
  int ihead = blockIdx.y/B;
  int ibatch = (blockIdx.y-ihead*B);
  int nW = (W-1)/stride0+1;
  int nHW = nW*((H-1)/stride0+1);

  // -- cuda threads --
  int pi = threadIdx.y;
  int pj = threadIdx.z;

  // -- across queries --
  for(int _qi = 0; _qi < q_per_thread; _qi++){

    // -- query index --
    qi = q_start + _qi;
    if (qi >= Q){ continue; }
    get_pixel_loc<int>(ref,qi,stride0,nW,nHW,H,W);

    // -- reference pixel index --
    ref_p[0] = ref[0];
    ref_p[1] = ref[1]+dilation*(pi + patch_offset);
    ref_p[2] = ref[2]+dilation*(pj + patch_offset);

    // -- valid ref pixel only --
    check_bounds(valid, ref_p, T,  H, W);
    if (not valid){ continue; }

    for(int _ki = 0; _ki < k_per_thread; _ki++){

      // -- non-local index --
      ki = k_start + _ki;
      if (ki >= K){ continue; }
  #pragma unroll
      for (int _idx=0; _idx < 3; _idx++){
        nl[_idx] = ref[_idx] + inds[ibatch][ihead][qi][ki][_idx];
      }

      // -- reflection with signs for backward step --
      int signH,signW;
      nl[0] = bounds(nl[0],T);
      signH = check_bound(nl[1],H) ? 1 : -1;
      nl[1] = bounds(nl[1],H);
      signW = check_bound(nl[2],W) ? 1 : -1;
      nl[2] = bounds(nl[2],W);

      // -- non-local pixel index --
      nl[1] = nl[1]+dilation*(pi + patch_offset);
      signH = check_bound(nl[1],H) ? signH : -signH;
      nl[1] = reflect_bounds ? bounds(nl[1],H) : nl[1];
      nl[2] = nl[2]+dilation*(pj + patch_offset);
      signW = check_bound(nl[2],W) ? signW : -signW;
      nl[2] = reflect_bounds ? bounds(nl[2],W) : nl[2];

      // -- valid non-local patches only --
      check_bounds(valid, nl, T,  H, W);
      if (not valid){ continue; }

      // -- non-local weight --
      weight = dists[ibatch][ihead][qi][ki];

      // -- gradient accumulation --
      scalar_t acc_dists_grad = 0;
      scalar_t acc_igradH = 0;
      scalar_t acc_igradW = 0;
      scalar_t igradH = 0;
      scalar_t igradW = 0;

      for (int pk = 0; pk < pt; pk++){

        // -- time is always valid --
        ref_ti = ref_p[0] + pk;
        nl_ti = reflect_bounds ? bounds(nl[0]+pk,T) : (nl[0]+pk);
        valid = check_bound(nl_ti, T) and check_bound(ref_ti, T);
        if (not valid){ continue; }
  
        // -- num features --
        for (int iftr = 0; iftr < F; iftr++){

          // -- read gradient --
          grad = out_vid_grad[ibatch][ihead][ref_ti][iftr][ref_p[1]][ref_p[2]];

          // -- write "in_vid_grad" and read "in_vid" @ non-local index --
          bilin2d_assign_bwd(igradW, igradH, pix_m,
                             weight*grad, nl[1], nl[2], H, W,
                             vid[ibatch][ihead][nl_ti][iftr],
                             in_vid_grad[ibatch][ihead][nl_ti][iftr]);

          // -- accumulate dists --
          acc_dists_grad += grad*pix_m;
          acc_igradW += grad*igradW;
          acc_igradH += grad*igradH;
        }

      } // pt

      // -- write dist grad --
      atomicAdd(&dists_grad[ibatch][ihead][qi][ki],acc_dists_grad);

      // -- write flows grad --
      atomicAdd(&inds_grad[ibatch][ihead][qi][ki][1],weight*acc_igradH*signH);
      atomicAdd(&inds_grad[ibatch][ihead][qi][ki][2],weight*acc_igradW*signW);


    } // ki
  } // qi
}

void wpsum_bilin2d_backward_cuda(
    torch::Tensor in_vid_grad,
    torch::Tensor dists_grad, torch::Tensor inds_grad,
    const torch::Tensor out_vid_grad, const torch::Tensor vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int stride0, int pt, int dilation, bool reflect_bounds, int patch_offset){

  // -- launch parameters --
  int B = dists.size(0);
  int HD = dists.size(1);
  int Q = dists.size(2);
  int K = dists.size(3);
  int q_per_thread = 1;
  int k_per_thread = K;
  // fprintf(stdout,
  //         "ps,stride0,pt,dilation,reflect_bounds,patch_offset: %d,%d,%d,%d,%d,%d\n",
  //         ps,stride0,pt,dilation,reflect_bounds,patch_offset);
  
  // -- kernel threads --
  int MAX_THREADS = 768;
  int q_threads = MAX_THREADS/(ps*ps); // num of queries threads per block
  q_threads = min(Q,q_threads);
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  int k_blocks = (K-1)/k_per_thread+1;
  dim3 nthreads(q_threads,ps,ps);
  dim3 nblocks(q_blocks, HD*B);

  // fprintf(stdout,"q_threads: %d\n",q_threads);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(in_vid_grad.type(), "wpsum_bilin2d_backward_vid_kernel",
                             ([&] {
    wpsum_bilin2d_backward_kernel<scalar_t><<<nblocks, nthreads>>>(
        in_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists_grad.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        out_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        ps, stride0, pt, dilation, reflect_bounds, patch_offset,
        q_per_thread, k_per_thread);
      }));
  
}

