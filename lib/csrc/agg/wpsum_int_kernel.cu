#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../shared_kernel.cu"

/****************************

       Forward Pass

****************************/

template <typename scalar_t>
__global__ void wpsum_int_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid,
    torch::PackedTensorAccessor32<int,6,torch::RestrictPtrTraits> counts,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> inds,
    int ps, int stride0, int pt, int dilation, bool reflect_bounds,
    int patch_offset, int q_per_thread){

    // -- shapes --
    int B = in_vid.size(0);
    int HD = in_vid.size(1);
    int T = in_vid.size(2);
    int F = in_vid.size(3);
    int H = in_vid.size(4);
    int W = in_vid.size(5);
    int Q = inds.size(2);
    int K = inds.size(3);

    // -- batching --
    int query_start = (threadIdx.x + blockDim.x*blockIdx.x)*q_per_thread;
    int ibatch = blockIdx.y;
    int ihead = blockIdx.z;

    // -- cuda threads --
    int pi = threadIdx.y;
    int pj = threadIdx.z;

    // -- pixel locations --
    int qi;
    bool valid;
    scalar_t pix,weight;
    int ref_ti,nl_ti;
    int ref[3];
    int nl[3];
    int nW = (W-1)/stride0+1;
    int nHW = nW*((H-1)/stride0+1);

    // -- across queries --
    for(int _qi = 0; _qi < q_per_thread; _qi++){

      // -- query index --
      qi = query_start + _qi;
      if (qi >= Q){ continue; }
      get_pixel_loc<int>(ref,qi,stride0,nW,nHW,H,W);

      // -- reference pixel index --
      ref[1] = ref[1]+dilation*(pi + patch_offset);
      ref[2] = ref[2]+dilation*(pj + patch_offset);

      // -- valid ref pixel only --
      valid = (ref[1] >= 0) && (ref[1] < H);
      valid = valid && (ref[2] >= 0) && (ref[2] < W);
      if (not valid){ continue; }

      for(int ki = 0; ki < K; ki++){

        // -- non-local index --
    #pragma unroll
        for (int _idx=0; _idx < 3; _idx++){
          nl[_idx] = ref[_idx] + inds[ibatch][ihead][qi][ki][_idx];
        }
  
        // -- always reflect anchor point --
        // nl[1] = bounds(nl[1],H);
        // nl[2] = bounds(nl[2],W);

        // -- non-local pixel index --
        nl[1] = nl[1]+dilation*(pi + patch_offset);
        nl[1] = reflect_bounds ? bounds(nl[1],H) : nl[1];
        nl[2] = nl[2]+dilation*(pj + patch_offset);
        nl[2] = reflect_bounds ? bounds(nl[2],W) : nl[2];

        // -- valid non-local patches only --
        valid = (nl[1] >= 0) && (nl[1] < H);
        valid = valid && (nl[2] >= 0) && (nl[2] < W);
        if (not valid){ continue; }

        // -- non-local weight --
        weight = dists[ibatch][ihead][qi][ki];

        // -- iterate over loop --
        for(int pk = 0; pk < pt; pk++){

          // -- time is always valid --
          ref_ti = bounds(ref[0] + pk,T);
          nl_ti = bounds(nl[0] + pk,T);

          // -- channels --
          for(int iftr = 0; iftr < F; iftr++){

            // -- fill --
            pix = weight*in_vid[ibatch][ihead][nl_ti][iftr][nl[1]][nl[2]];
            atomicAdd(&out_vid[ibatch][ihead][ref_ti][iftr][ref[1]][ref[2]],pix);

          } // nfeatures-loop
        } // pt-loop
      } // k-loop

      // -- normalize --
      if ((ref[0]==0) and (ibatch==0) and (ihead==0)){
        atomicAdd(&counts[0][0][0][0][ref[1]][ref[2]],1);
      }
      // atomicAdd(&counts[0][0][0][0][ref[1]][ref[2]],1);

    } // query-loop
}

void wpsum_int_forward_cuda(
    torch::Tensor out_vid, torch::Tensor counts,
    const torch::Tensor in_vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int stride0, int pt, int dilation,
    bool reflect_bounds, int patch_offset){

  // -- unpack --
  int B = inds.size(0);
  int HD = inds.size(1);
  int Q = inds.size(2);
  int q_per_thread = 2;

  // -- kernel threads --
  int MAX_THREADS = 1024;
  int q_threads = MAX_THREADS/(ps*ps); // num of queries threads per block
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  dim3 nthreads(q_threads,ps,ps);

  // -- kernel blocks --
  dim3 nblocks(q_blocks,B,HD);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(in_vid.type(), "wpsum_int_forward_kernel", ([&] {
    wpsum_int_forward_kernel<scalar_t><<<nblocks, nthreads>>>(
        out_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        counts.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
        in_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
        ps, stride0, pt, dilation, reflect_bounds, patch_offset,
        q_per_thread);
    }));
}



/************************************

  Backward Pass (for Vid & Dists)

*************************************/

template <typename scalar_t>
__global__ void wpsum_int_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> inds,
    int ps, int stride0, int pt, int dilation, bool reflect_bounds, int patch_offset,
    int q_per_thread, int k_per_thread){

  // -- shape --
  int B =  dists.size(0);
  int HD = dists.size(1);
  int Q =  dists.size(2);
  int K =  dists.size(3);
  int T = out_vid_grad.size(2);
  int F = out_vid_grad.size(3);
  int H = out_vid_grad.size(4);
  int W = out_vid_grad.size(5);

  // -- pixel indexing --
  int qi,ki;
  int ref[3];
  scalar_t nl[3];
  int ref_ti,nl_ti;
  bool valid;
  float weight,pix_n,pix_m;

  // -- location to fill --
  int q_start = blockIdx.x*blockDim.x+threadIdx.x;
  int k_start = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/B;
  int ibatch = (blockIdx.z-ihead*B) % B;
  int nW = (W-1)/stride0+1;
  int nHW = nW*((H-1)/stride0+1);

  // -- cuda threads --
  int pi = threadIdx.y;
  int pj = threadIdx.z;

  // -- across queries --
  for(int _qi = 0; _qi < q_per_thread; _qi++){

    // -- query index --
    qi = q_start + _qi;
    if (qi >= Q){ continue; }
    get_pixel_loc<int>(ref,qi,stride0,nW,nHW,H,W);

    // -- reference pixel index --
    ref[1] = ref[1]+dilation*(pi + patch_offset);
    ref[2] = ref[2]+dilation*(pj + patch_offset);

    // -- valid ref pixel only --
    valid = (ref[1] >= 0) && (ref[1] < H);
    valid = valid && (ref[2] >= 0) && (ref[2] < W);
    if (not valid){ continue; }

    for(int _ki = 0; _ki < k_per_thread; _ki++){

      // -- non-local index --
      ki = k_start + _ki;
      if (ki >= K){ continue; }
  #pragma unroll
      for (int _idx=0; _idx < 3; _idx++){
        nl[_idx] = ref[_idx] + inds[ibatch][ihead][qi][ki][_idx];
      }

      // -- non-local pixel index --
      nl[1] = nl[1]+dilation*(pi + patch_offset);
      nl[1] = reflect_bounds ? bounds(nl[1],H) : nl[1];
      nl[2] = nl[2]+dilation*(pj + patch_offset);
      nl[2] = reflect_bounds ? bounds(nl[2],W) : nl[2];

      // -- valid non-local patches only --
      valid = (nl[1] >= 0) && (nl[1] < H);
      valid = valid && (nl[2] >= 0) && (nl[2] < W);
      if (not valid){ continue; }

      // -- non-local weight --
      weight = dists[ibatch][ihead][qi][ki];
      scalar_t acc_dists_grad = 0;

      for (int pk = 0; pk < pt; pk++){
        ref_ti = ref[0] + pk;
        nl_ti = nl[0] + pk;
  
        // -- num features --
        for (int iftr = 0; iftr < F; iftr++){
          pix_n = out_vid_grad[ibatch][ihead][ref_ti][iftr][ref[1]][ref[2]];
          pix_m = vid[ibatch][ihead][nl_ti][iftr][nl[1]][nl[2]];
          atomicAdd(&in_vid_grad[ibatch][ihead][nl_ti][iftr][nl[1]][nl[2]],weight*pix_n);
          acc_dists_grad += pix_n*pix_m;
        }

      } // pt

      // -- write dist grad --
      atomicAdd(&dists_grad[ibatch][ihead][qi][ki],acc_dists_grad);

    } // ki
  } // qi
}

void wpsum_int_backward_cuda(
    torch::Tensor in_vid_grad, torch::Tensor dists_grad,
    const torch::Tensor out_vid_grad, const torch::Tensor vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int stride0, int pt, int dilation, bool reflect_bounds, int patch_offset){

  // -- launch parameters --
  int B = dists.size(0);
  int HD = dists.size(1);
  int Q = dists.size(2);
  int K = dists.size(3);
  int q_per_thread = 2;
  int k_per_thread = K;
  
  // -- kernel threads --
  int MAX_THREADS = 1024;
  int q_threads = MAX_THREADS/(ps*ps); // num of queries threads per block
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  int k_blocks = (K-1)/k_per_thread+1;
  dim3 nthreads(q_threads,ps,ps);
  dim3 nblocks(q_blocks, k_blocks, HD*B);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(in_vid_grad.type(), "wpsum_int_backward_vid_kernel", ([&] {
    wpsum_int_backward_kernel<scalar_t><<<nblocks, nthreads>>>(
        in_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists_grad.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        out_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
        ps, stride0, pt, dilation, reflect_bounds, patch_offset,
        q_per_thread, k_per_thread);
      }));
  
}

