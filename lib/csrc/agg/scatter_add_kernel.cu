#include "hip/hip_runtime.h"
/*

  Stack non-local patches into a video

*/

// #include "scatter_int.cu"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../shared_kernel.cu"

/****************************

       Forward Pass

****************************/

template <typename scalar_t, typename itype, bool INTERPOLATE>
__global__ void scatter_add_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid,
    torch::PackedTensorAccessor32<int,2,torch::RestrictPtrTraits> counts,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<itype,5,torch::RestrictPtrTraits> inds,
    int ps, int strideIn, int strideOut, int pt, int dilation, bool reflect_bounds,
    int patch_offset, int q_per_thread){

    // -- shapes --
    int B = in_vid.size(0);
    int HD = in_vid.size(1);
    int T = in_vid.size(2);
    int F = in_vid.size(3);
    int inH = in_vid.size(4);
    int inW = in_vid.size(5);
    int outH = out_vid.size(4);
    int outW = out_vid.size(5);
    int Q = inds.size(2);
    int K = inds.size(3);

    // -- batching --
    int query_start = q_per_thread*(threadIdx.x + blockDim.x*blockIdx.x);
    int ki = blockIdx.y*blockDim.y+threadIdx.y;
    int ihead = blockIdx.z/B;
    int ibatch = (blockIdx.z-ihead*B) % B;
    if (ki >= K){ return; }

    // -- pixel locations --
    int qi;
    bool valid;
    scalar_t pix,weight;
    int nl_ti,ref[3],ref_p[3];
    itype nl[3],nl_p[3];
    int nW = (inW-1)/strideIn+1;
    int nHW = nW*((inH-1)/strideIn+1);

    // -- across queries --
    for(int _qi = 0; _qi < q_per_thread; _qi++){

      // -- query index --
      qi = query_start + _qi;
      if (qi >= Q){ continue; }

      // -- write location --
      get_pixel_loc(ref,qi,strideIn,nW,nHW,inH,inW);

      // -- non-local index --
      get_pixel_loc(nl,qi,strideOut,nW,nHW,outH,outW);
  #pragma unroll
      for (int _idx=0; _idx < 3; _idx++){
        nl[_idx] = nl[_idx] + inds[ibatch][ihead][qi][ki][_idx];
      }

      // -- check "inf" (but it won't be inf sometimes)  --
      valid = (abs(nl[1]) < 1e7) and (abs(nl[2]) < 1e7);
      if (not(valid)){ continue; }

      // -- always reflect anchor point --
      nl[0] = bounds(nl[0],T);
      nl[1] = bounds(nl[1],outH);
      nl[2] = bounds(nl[2],outW);

      // -- non-local weight --
      weight = dists[ibatch][ihead][qi][ki];

      // -- iterate over patches --
      for(int pi=0; pi < ps; pi++){
      for(int pj=0; pj < ps; pj++){

        // -- reference pixel index --
        ref_p[0] = ref[0];
        ref_p[1] = ref[1]+dilation*(pi + patch_offset);
        ref_p[2] = ref[2]+dilation*(pj + patch_offset);
        check_bounds(valid, ref_p, T,  inH, inW);
        if (not valid){ continue; }
  
        // -- increment legal refs --
        if ((ref[0]==0) and (ibatch==0) and (ihead==0) and (ki==0)){
          atomicAdd(&counts[ref_p[1]][ref_p[2]],1);
        }
  
        // -- non-local pixel index --
        nl_p[0] = nl[0];
        nl_p[1] = nl[1]+dilation*(pi + patch_offset);
        nl_p[1] = reflect_bounds ? bounds(nl_p[1],inH) : nl_p[1];
        nl_p[2] = nl[2]+dilation*(pj + patch_offset);
        nl_p[2] = reflect_bounds ? bounds(nl_p[2],inW) : nl_p[2];
        check_bounds(valid, nl_p, T, inH, inW);
        if (not valid){ continue; }

        // -- iterate over loop --
        for(int pk = 0; pk < pt; pk++){

          // -- time is always valid --
          ref_p[0] = ref[0] + pk;
          nl_p[0] = reflect_bounds ? bounds(nl[0]+pk,T) : (nl[0]+pk);
          nl_ti = nl_p[0];
          valid = (nl_p[0] >= 0) and (nl_p[0] < T) and (ref_p[0] >= 0) and (ref_p[0] < T);
          if (not valid){ continue; }

          // -- channels --
          for(int iftr = 0; iftr < F; iftr++){

            // -- read --
            pix = weight*in_vid[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]];
            // atomicAdd(&out_vid[ibatch][ihead][nl_ti][iftr][nl[1]][nl[2]],pix);

            // -- write --
            if (INTERPOLATE){
              bilin2d_assign(pix,(scalar_t)nl_p[1],(scalar_t)nl_p[2],inH,inW,
                             out_vid[ibatch][ihead][nl_ti][iftr]);
            }else{
              atomicAdd(&out_vid[ibatch][ihead][nl_p[0]][iftr][nl_p[1]][nl_p[2]],pix);
            }


          } // nfeatures-loop
        } // pt-loop
      }} // pi,pj
  } // query-loop
}

void scatter_add_forward_cuda(
    torch::Tensor out_vid, torch::Tensor counts,
    const torch::Tensor in_vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int strideIn, int strideOut, int pt,
    int dilation, bool reflect_bounds, int patch_offset, bool itype_int){

  // -- unpack --
  int B = inds.size(0);
  int HD = inds.size(1);
  int Q = inds.size(2);
  int K = inds.size(3);
  int q_per_thread = 2;

  // -- check dims --
  int inH = in_vid.size(4);
  int inW = in_vid.size(5);
  int outH = out_vid.size(4);
  int outW = out_vid.size(5);
  assert(inH <= outH);
  assert(inW <= outW);

  // -- kernel threads --
  int MAX_THREADS = 512;//1024
  int k_threads = 8;
  int q_threads = MAX_THREADS/(k_threads); // num of queries threads per block
  q_threads = min(Q,q_threads);
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  int k_blocks = (K-1)/(k_threads)+1;
  dim3 nthreads(q_threads,k_threads);

  // -- kernel blocks --
  dim3 nblocks(q_blocks,k_blocks,B*HD);

  // -- launch kernel --
  if (itype_int){
    AT_DISPATCH_FLOATING_TYPES(in_vid.type(),
                               "scatter_add_int_forward_kernel", ([&] {
    scatter_add_forward_kernel<scalar_t,int,true><<<nblocks, nthreads>>>(
          out_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          counts.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
          in_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          ps, strideIn, strideOut, pt, dilation, reflect_bounds, patch_offset,
          q_per_thread);
      }));
  }else{
    AT_DISPATCH_FLOATING_TYPES(in_vid.type(),
                               "scatter_add_bilin2d_forward_kernel", ([&] {
    scatter_add_forward_kernel<scalar_t,scalar_t,true><<<nblocks, nthreads>>>(
          out_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          counts.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
          in_vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          ps, strideIn, strideOut, pt, dilation, reflect_bounds, patch_offset,
          q_per_thread);
      }));
  }
}



/************************************

  Backward Pass (for Vid & Dists)

*************************************/

template <typename scalar_t>
__global__ void scatter_add_int_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> inds,
    int ps, int strideIn, int strideOut, int pt, int dilation,
    bool reflect_bounds, int patch_offset, int q_per_thread){

  // -- shape --
  int B =  dists.size(0);
  int HD = dists.size(1);
  int Q =  dists.size(2);
  int K =  dists.size(3);
  int T = out_vid_grad.size(2);
  int F = out_vid_grad.size(3);
  int inH = in_vid_grad.size(4);
  int inW = in_vid_grad.size(5);
  int outH = out_vid_grad.size(4);
  int outW = out_vid_grad.size(5);

  // -- pixel indexing --
  int qi;
  int ref[3],ref_p[3],nl[3],nl_p[3];
  bool valid;
  float weight,grad,pix_m;

  // -- batching --
  int query_start = q_per_thread*(threadIdx.x + blockDim.x*blockIdx.x);
  int ki = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/B;
  int ibatch = (blockIdx.z-ihead*B) % B;
  if (ki >= K){ return; }
  int nW = (outW-1)/strideOut+1;
  int nHW = nW*((outH-1)/strideOut+1);

  // -- across queries --
  for(int _qi = 0; _qi < q_per_thread; _qi++){

    // -- query index --
    qi = query_start + _qi;
    if (qi >= Q){ continue; }
    get_pixel_loc<int>(ref,qi,strideOut,nW,nHW,inH,inW);

    // -- non-local index --
#pragma unroll
    get_pixel_loc<int>(nl,qi,strideIn,nW,nHW,outH,outW);
    for (int _idx=0; _idx < 3; _idx++){
      nl[_idx] = nl[_idx] + inds[ibatch][ihead][qi][ki][_idx];
    }

    // -- check "inf" (but it won't be inf sometimes)  --
    valid = (abs(nl[1]) < 1e7) and (abs(nl[2]) < 1e7);
    if (not(valid)){ continue; }

    // -- always reflect anchor point --
    nl[0] = bounds(nl[0],T);
    nl[1] = bounds(nl[1],outH);
    nl[2] = bounds(nl[2],outW);

    // -- non-local weight --
    weight = dists[ibatch][ihead][qi][ki];

    // -- iterate over patches --
    for(int pi=0; pi < ps; pi++){
    for(int pj=0; pj < ps; pj++){

        // -- reference pixel index --
        ref_p[0] = ref[0];
        ref_p[1] = ref[1]+dilation*(pi + patch_offset);
        ref_p[2] = ref[2]+dilation*(pj + patch_offset);
        check_bounds(valid, ref_p, T, inH, inW);
        if (not valid){ continue; }
  
        // // -- increment legal refs --
        // if ((ref_p[0]==0) and (ibatch==0) and (ihead==0) and (ki==0)){
        //   atomicAdd(&counts[ref_p[1]][ref_p[2]],1);
        // }
  
        // -- non-local pixel index --
        nl_p[0] = nl[0];
        nl_p[1] = nl[1]+dilation*(pi + patch_offset);
        nl_p[1] = reflect_bounds ? bounds(nl_p[1],outH) : nl_p[1];
        nl_p[2] = nl[2]+dilation*(pj + patch_offset);
        nl_p[2] = reflect_bounds ? bounds(nl_p[2],outW) : nl_p[2];
        check_bounds(valid, nl_p, T, outH, outW);
        if (not valid){ continue; }

        // -- init accumulation --
        scalar_t acc_dists_grad = 0;

        for (int pk = 0; pk < pt; pk++){

          // -- time is always valid --
          ref_p[0] = ref[0] + pk;
          nl_p[0] = reflect_bounds ? bounds(nl[0]+pk,T) : (nl[0]+pk);
          valid = (nl_p[0] >= 0) and (nl_p[0] < T) and (ref_p[0] >= 0) and (ref_p[0] < T);
          if (not valid){ continue; }

          // -- num features --
          for (int iftr = 0; iftr < F; iftr++){
            grad = out_vid_grad[ibatch][ihead][nl_p[0]][iftr][nl_p[1]][nl_p[2]];
            pix_m = vid[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]];
            atomicAdd(&in_vid_grad[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]],
                      weight*grad);
            acc_dists_grad += grad*pix_m;
          }

        } // pt

      // -- write dist grad --
      atomicAdd(&dists_grad[ibatch][ihead][qi][ki],acc_dists_grad);

    }} // pi,pj
  } // qi
}

void scatter_add_int_backward_cuda(
    torch::Tensor in_vid_grad, torch::Tensor dists_grad,
    const torch::Tensor out_vid_grad, const torch::Tensor vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int strideIn, int strideOut, int pt, int dilation,
    bool reflect_bounds, int patch_offset){

  // -- launch parameters --
  int B = dists.size(0);
  int HD = dists.size(1);
  int Q = dists.size(2);
  int K = dists.size(3);
  int q_per_thread = 1;
  
  // -- kernel threads --
  int MAX_THREADS = 512;
  int k_threads = 8;
  int q_threads = MAX_THREADS/(k_threads); // num of queries threads per block
  q_threads = min(Q,q_threads);
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  int k_blocks = (K-1)/(k_threads)+1;
  dim3 nthreads(q_threads,k_threads);

  // -- kernel blocks --
  dim3 nblocks(q_blocks,k_blocks,B*HD);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(in_vid_grad.type(),
                             "scatter_add_int_backward_vid_kernel", ([&] {
    scatter_add_int_backward_kernel<scalar_t><<<nblocks, nthreads>>>(
        in_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists_grad.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        out_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
        ps, strideIn, strideOut, pt, dilation, reflect_bounds, patch_offset,
        q_per_thread);
      }));
  
}

/************************************

  Bilin2d Backward Pass (for Vid & Dists)

*************************************/

template <typename scalar_t>
__global__ void scatter_add_bilin2d_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> in_vid_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> out_vid_grad,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> vid,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds,
    int ps, int strideIn, int strideOut, int pt, int dilation,
    bool reflect_bounds, int patch_offset, int q_per_thread){


  // -- shape --
  int B =  dists.size(0);
  int HD = dists.size(1);
  int Q =  dists.size(2);
  int K =  dists.size(3);
  int T = out_vid_grad.size(2);
  int F = out_vid_grad.size(3);
  int inH = in_vid_grad.size(4);
  int inW = in_vid_grad.size(5);
  int outH = out_vid_grad.size(4);
  int outW = out_vid_grad.size(5);

  // -- pixel indexing --
  bool valid;
  int qi;
  scalar_t weight,grad,pix_m;
  int ref[3],ref_p[3],nl_ti;
  scalar_t nl[3],nl_p[3];

  // -- batching --
  int query_start = q_per_thread*(threadIdx.x + blockDim.x*blockIdx.x);
  int ki = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/B;
  int ibatch = (blockIdx.z-ihead*B) % B;
  if (ki >= K){ return; }
  int nW = (inW-1)/strideIn+1;
  int nHW = nW*((inH-1)/strideIn+1);

  // -- across queries --
  for(int _qi = 0; _qi < q_per_thread; _qi++){

    // -- query index --
    qi = query_start + _qi;
    if (qi >= Q){ continue; }
    get_pixel_loc(ref,qi,strideOut,nW,nHW,inH,inW);

    // -- non-local index --
#pragma unroll
    get_pixel_loc(nl,qi,strideIn,nW,nHW,outH,outW);
    for (int _idx=0; _idx < 3; _idx++){
      nl[_idx] = nl[_idx] + inds[ibatch][ihead][qi][ki][_idx];
    }

    // -- check "inf" (but it won't be inf sometimes)  --
    valid = (abs(nl[1]) < 1e7) and (abs(nl[2]) < 1e7);
    if (not(valid)){ continue; }

    // -- always reflect anchor point --
    int signH0,signW0;
    nl[0] = bounds(nl[0],T);
    signH0 = check_bound(nl[1],outH) ? 1 : -1;
    nl[1] = bounds(nl[1],outH);
    signW0 = check_bound(nl[2],outW) ? 1 : -1;
    nl[2] = bounds(nl[2],outW);

    // -- non-local weight --
    weight = dists[ibatch][ihead][qi][ki];

    // -- iterate over patches --
    for(int pi=0; pi < ps; pi++){
    for(int pj=0; pj < ps; pj++){

        // -- reference pixel index --
        ref_p[0] = ref[0];
        ref_p[1] = ref[1]+dilation*(pi + patch_offset);
        ref_p[2] = ref[2]+dilation*(pj + patch_offset);
        check_bounds(valid, ref_p, T, inH, inW);
        if (not valid){ continue; }
  
        // // -- increment legal refs --
        // if ((ref_p[0]==0) and (ibatch==0) and (ihead==0) and (ki==0)){
        //   atomicAdd(&counts[ref_p[1]][ref_p[2]],1);
        // }
  
        // -- non-local pixel index --
        nl_p[0] = nl[0];
        nl_p[1] = nl[1]+dilation*(pi + patch_offset);
        int signH = check_bound(nl_p[1],outH) ? signH0 : -signH0;
        nl_p[1] = reflect_bounds ? bounds(nl_p[1],outH) : nl_p[1];
        nl_p[2] = nl[2]+dilation*(pj + patch_offset);
        int signW = check_bound(nl_p[2],outW) ? signW0 : -signW0;
        nl_p[2] = reflect_bounds ? bounds(nl_p[2],outW) : nl_p[2];
        check_bounds(valid, nl_p, T, outH, outW);
        if (not valid){ continue; }

        // -- gradient accumulation --
        scalar_t acc_dists_grad = 0;
        scalar_t acc_igradH = 0;
        scalar_t acc_igradW = 0;
        scalar_t igradH = 0;
        scalar_t igradW = 0;

        // -- time patch --
        for (int pk = 0; pk < pt; pk++){

          // -- time is always valid --
          ref_p[0] = ref_p[0] + pk;
          nl_ti = reflect_bounds ? bounds(nl[0]+pk,T) : (nl[0]+pk);
          valid = (nl_p[0] >= 0) and (nl_p[0] < T);
          valid = valid and (ref_p[0] >= 0) and (ref_p[0] < T);
          if (not valid){ continue; }

          // -- num features --
          for (int iftr = 0; iftr < F; iftr++){

            // -- read gradient --
            // grad = out_vid_grad[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]];
            // bilin2d_interpolate(grad,(scalar_t)nl_p[1],(scalar_t)nl_p[2],inH,inW,
            //                     out_vid_grad[ibatch][ihead][nl_ti][iftr]);

            // bilin2d_read_bwd(igradW, igradH, pix_m, grad,
            //                  nl_p[1], nl_p[2], outH, outW,
            //                  out_vid_grad[ibatch][ihead][nl_ti][iftr]);
            pix_m = vid[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]];
            // bilin2d_assign_bwd(igradW, igradH, pix_m,
            //                    weight*grad, nl_p[1], nl_p[2], inH, inW,
            //                    vid[ibatch][ihead][nl_ti][iftr],
            //                    in_vid_grad[ibatch][ihead][nl_ti][iftr]);


            // grad = out_vid_grad[ibatch][ihead][nl_p[0]][iftr][nl_p[1]][nl_p[2]];
            // pix_m = vid[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]];
            // atomicAdd(&in_vid_grad[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]],
            //           weight*grad);
            // acc_dists_grad += grad*pix_m;

            // -- write at in_vid_grad
            atomicAdd(&in_vid_grad[ibatch][ihead][ref_p[0]][iftr][ref_p[1]][ref_p[2]],
                      weight*pix_m);

            // -- accumulate dists --
            acc_dists_grad += grad*pix_m;
            acc_igradW += grad*igradW;
            acc_igradH += grad*igradH;
          }

        } // pt

      // -- write dist grad --
      atomicAdd(&dists_grad[ibatch][ihead][qi][ki],acc_dists_grad);

      // -- write flows grad --
      atomicAdd(&inds_grad[ibatch][ihead][qi][ki][1],weight*acc_igradH*signH);
      atomicAdd(&inds_grad[ibatch][ihead][qi][ki][2],weight*acc_igradW*signW);

    }} // pi,pj
  } // qi

}

void scatter_add_bilin2d_backward_cuda(
    torch::Tensor in_vid_grad,
    torch::Tensor dists_grad, torch::Tensor inds_grad,
    const torch::Tensor out_vid_grad, const torch::Tensor vid,
    const torch::Tensor dists, const torch::Tensor inds,
    int ps, int strideIn, int strideOut, int pt, int dilation,
    bool reflect_bounds, int patch_offset){

  // -- launch parameters --
  int B = dists.size(0);
  int HD = dists.size(1);
  int Q = dists.size(2);
  int K = dists.size(3);
  int q_per_thread = 2;
  
  // -- kernel threads --
  int MAX_THREADS = 512;
  int k_threads = 8;
  int q_threads = MAX_THREADS/(k_threads); // num of queries threads per block
  q_threads = min(Q,q_threads);
  int q_blocks = (Q-1)/(q_per_thread*q_threads)+1;
  int k_blocks = (K-1)/(k_threads)+1;
  dim3 nthreads(q_threads,k_threads);

  // -- kernel blocks --
  dim3 nblocks(q_blocks,k_blocks,B*HD);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(in_vid_grad.type(),
                             "scatter_add_bilin2d_backward_vid_kernel", ([&] {
    scatter_add_bilin2d_backward_kernel<scalar_t><<<nblocks, nthreads>>>(
        in_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists_grad.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        out_vid_grad.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        vid.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        ps, strideIn, strideOut, pt, dilation, reflect_bounds, patch_offset,
        q_per_thread);
      }));
  
}
