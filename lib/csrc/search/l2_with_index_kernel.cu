#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

/****************************

       Inline Functions

****************************/

__inline__ __device__ int bounds(int val, int lim ){
  int vval = val;
  if (val < 0){
    vval = -val;
  }else if (val >= lim){
    vval = 2*(lim-1) - val;
  }
  return vval;
}

inline __host__ __device__
int unravel_index(int& ti, int& hi, int& wi, const int qindex,
                  const int h, const int w, const int hw){
  // index to pixel location
  int i_mod = qindex % hw;
  ti = qindex / hw;
  wi = (i_mod % w);
  hi = (i_mod / w) % h;
}

/****************************

       Forward Pass

****************************/

template <typename scalar_t>
__global__ void l2_search_with_index_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid0,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid1,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fflow,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> bflow,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> inds,
    int qstart, int nqueries, int stride0, int n_h0, int n_w0,
    int h0_off, int w0_off, int h1_off, int w1_off,
    int ps, int pt, int ws_h, int ws_w, int wt,
    int chnls, int dilation, int stride1,
    bool use_adj, bool reflect_bounds, bool search_abs, bool full_ws,
    torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> bufs,
    torch::PackedTensorAccessor32<int,2,torch::RestrictPtrTraits> tranges,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> n_tranges,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> min_tranges,
    int ws_h_iters, int ws_w_iters, int bpb){

  // shapes
  int nframes,color,h,w,height,width;
  nframes = vid0.size(0);
  color = vid0.size(1);
  h = vid0.size(2);
  w = vid0.size(3);
  height = h;
  width = w;
  int n_hw0 = n_h0 * n_w0;

  // constants
  float nan = __int_as_float(0xffe00000);
  float inf = __int_as_float(0x7f800000);

  // offsets
  int psHalf = (ps)/2;
  int wsHalf_h = (ws_h)/2;
  int wsHalf_w = (ws_w)/2;
  int wsMax_h = stride1*(ws_h-1-wsHalf_h);
  int wsMax_w = stride1*(ws_w-1-wsHalf_w);
  int adj = use_adj ? psHalf : 0;

  // column index
  int blkDimX = blockDim.x; // num threads in x-block
  int blkDimY = blockDim.y; // num threads in y-block
  int cu_tidX = threadIdx.x;
  int cu_tidY = threadIdx.y;
  int block_start = blockIdx.x*bpb;
  int bidx,ws_i,ws_j,dtd;

  // decls
  int ti,hi,wi;
  int n_ti,n_hi,n_wi;
  int vH,vW,vT,nH,nW,nT;
  bool valid,vvalid,nvalid;
  bool valid_ti,valid_hi,valid_wi,valid_anchor;
  bool valid_n_ti,valid_n_hi,valid_n_wi,valid_n;
  bool eq_ti,eq_hi,eq_wi,eq_dim;
  int wsOff_h,wsOff_w;

  float cw0,ch0,ct0,cw_f,ch_f;
  int l_cw0,l_ch0,l_ct0;
  int cw_i,ch_i,ch,cw,ct;
  float dist,v_pix,n_pix;

  for (int _bidx = 0; _bidx < bpb; _bidx++){

    //---------------------------
    //   extract anchor pixel
    //---------------------------

    // -- block start --
    bidx = block_start + _bidx;
    if (bidx >= nqueries){ continue; }

    // -- unpack pixel locs --
    int qindex = bidx + qstart;
    int i_mod = qindex % n_hw0;
    ti = qindex / n_hw0;
    wi = ((i_mod % n_w0) * stride0) % width ;
    hi = ((i_mod / n_w0) * stride0) % height;

    // unravel_index(ti, hi, wi, qindex, height, width, hw);

    // -- valid (anchor pixel) --
    valid_ti = (ti < nframes) && (ti >= 0);
    valid_hi = (hi < height) && (hi >= 0);
    valid_wi = (wi < width) && (wi >= 0);
    valid_anchor = valid_ti && valid_hi && valid_wi;

    // -- search offset --
    if(full_ws){
      wsOff_h = (hi-max(hi-stride1*wsHalf_h,0))/stride1;
      wsOff_w = (wi-max(wi-stride1*wsHalf_w,0))/stride1;
      if ((hi+wsMax_h) >= height){
        wsOff_h+=(hi+wsMax_h-min(hi+stride1*wsMax_h,height-1)-1)/stride1 + 1;
      }
      if ((wi+wsMax_w) >= width){
        wsOff_w+=(wi+wsMax_w-min(wi+stride1*wsMax_w,width-1)-1)/stride1 + 1;
      }
    }else{
      wsOff_h = wsHalf_h;
      wsOff_w = wsHalf_w;
    }

    // ---------------------------------------
    //     searching loop for (ti,top,left)
    // ---------------------------------------

    // -- we loop over search space if needed --
    for (int _xi = 0; _xi < ws_h_iters; _xi++){

      int ws_i = cu_tidX + blkDimX*_xi;
      if (ws_i >= ws_h){ continue; }

      for (int _yi = 0; _yi < ws_w_iters; _yi++){
        ws_j = cu_tidY + blkDimY*_yi;
        if (ws_j >= ws_w){ continue; }

        for( int wt_k = 0; wt_k < n_tranges[ti]; wt_k++){
          int n_ti = tranges[ti][wt_k];
          int dt = n_ti - min_tranges[ti];

          // ------------------------
          //      init direction
          // ------------------------
          int direction = max(-1,min(1,n_ti - ti));
          if (direction != 0){

            // -- get offset at index --
            int dtd = int(dt-direction);
            cw0 = 1.*bufs[bidx][0][dtd][ws_i][ws_j];
            ch0 = 1.*bufs[bidx][1][dtd][ws_i][ws_j];
            ct0 = 1.*bufs[bidx][2][dtd][ws_i][ws_j];

            // -- legalize access --
            l_cw0 = int(max(0,min(w-1,int(cw0))));
            l_ch0 = int(max(0,min(h-1,int(ch0))));
            l_ct0 = int(max(0,min(nframes-1,int(ct0))));

            // -- access flows --
            if (direction > 0 ){
              cw_f = cw0 + fflow[l_ct0][0][l_ch0][l_cw0];
              ch_f = ch0 + fflow[l_ct0][1][l_ch0][l_cw0];
            }else{
              cw_f = cw0 + bflow[l_ct0][0][l_ch0][l_cw0];
              ch_f = ch0 + bflow[l_ct0][1][l_ch0][l_cw0];
            }
            cw_i = int(cw_f+0.5);
            ch_i = int(ch_f+0.5);

            // -- rounding --
            cw = max(0,min(width-1,cw_i));
            ch = max(0,min(height-1,ch_i));
            ct = n_ti;

          }else{
            cw = wi;
            ch = hi;
            ct = ti;
          }

          
          // ----------------
          //     update
          // ----------------
          if (wt > 0){
            bufs[bidx][0][dt][ws_i][ws_j] = cw;
            bufs[bidx][1][dt][ws_i][ws_j] = ch;
            bufs[bidx][2][dt][ws_i][ws_j] = ct;
          }
          // cw = wi;
          // ch = hi;
          // ct = n_ti;

          // --------------------
          //      init dists
          // --------------------
          dist = 0;

          // -----------------
          //    spatial dir
          // -----------------
          if (search_abs){
            n_hi = stride1 * ws_i;
            n_wi = stride1 * ws_j;
          }else{
            n_hi = ch + stride1 * (ws_i - wsOff_h);
            n_wi = cw + stride1 * (ws_j - wsOff_w);
          }

          // ---------------------------
          //      valid (search "n")
          // ---------------------------
          valid_n_ti = (n_ti < nframes) && (n_ti >= 0);
          valid_n_hi = (n_hi < height) && (n_hi >= 0);
          valid_n_wi = (n_wi < width) && (n_wi >= 0);
          valid_n = valid_n_ti && valid_n_hi && valid_n_wi;
          valid = valid_n && valid_anchor;

          // ---------------------------------
          //
          //  compute delta over patch vol.
          //
          // ---------------------------------
          for (int pk = 0; pk < pt; pk++){
            for (int pi = 0; pi < ps; pi++){
              for (int pj = 0; pj < ps; pj++){
                
                // -- inside entire image --
                vH = (hi - h0_off) + dilation*(pi - psHalf + adj);
                vW = (wi - w0_off) + dilation*(pj - psHalf + adj);
                vH = reflect_bounds ? bounds(vH,height) : vH;
                vW = reflect_bounds ? bounds(vW,width)  : vW;
                vT = ti + pk;

                nH = (n_hi - h1_off) + dilation*(pi - psHalf + adj);
                nW = (n_wi - w1_off) + dilation*(pj - psHalf + adj);
                nH = reflect_bounds ? bounds(nH,height) : nH;
                nW = reflect_bounds ? bounds(nW,width)  : nW;
                nT = n_ti + pk;

                // -- valid checks [for testing w/ zero pads] --
                vvalid = (vH < height) && (vH >= 0);
                vvalid = vvalid && (vW < width) && (vW >= 0);
                vvalid = vvalid && (vT < nframes) && (vT >= 0);

                nvalid = (nH < height) && (nH >= 0);
                nvalid = nvalid && (nW < width) && (nW >= 0);
                nvalid = nvalid && (nT < nframes) && (nT >= 0);

                // -- all channels --
                for (int ci = 0; ci < chnls; ci++){

                  // -- get data --
                  if (vvalid){
                    v_pix = vid0[vT][ci][vH][vW];
                  }else{
                    v_pix = 0;
                  }

                  if (nvalid){
                    n_pix = vid1[nT][ci][nH][nW];
                  }else{
                    n_pix = 0;
                  }

                  // -- compute dist --
                  if (valid){
                    float _dist = (v_pix - n_pix);
                    dist += _dist*_dist;
                    // dist += v_pix * n_pix;
                  }
                }
              }
            }
          }

          // -- dists --
          if (!valid){ dist = inf; }
          dists[bidx][wt_k][ws_i][ws_j] = dist;

          // -- inds --
          inds[bidx][wt_k][ws_i][ws_j][0] = n_ti;
          inds[bidx][wt_k][ws_i][ws_j][1] = n_hi;
          inds[bidx][wt_k][ws_i][ws_j][2] = n_wi;

          // -- final check [put self@index 0] --
          // eq_ti = n_ti == ti;
          // eq_hi = n_hi == hi;
          // eq_wi = n_wi == wi;
          // eq_dim = eq_ti && eq_hi && eq_wi;
          // dist = dists[bidx][wt_k][ws_i][ws_j];
          // if (eq_dim){
          //   dists[bidx][wt_k][ws_i][ws_j] = -100;
          // }

        }
      }
    }
  }
}

void l2_search_with_index_forward_cuda(
    torch::Tensor vid0, torch::Tensor vid1,
    torch::Tensor fflow, torch::Tensor bflow,
    torch::Tensor dists, torch::Tensor inds,
    int qstart, int nqueries, int stride0, int n_h0, int n_w0,
    int h0_off, int w0_off, int h1_off, int w1_off,
    int ps, int pt, int ws_h, int ws_w, int wt,
    int chnls, int dilation, int stride1,
    bool use_adj, bool reflect_bounds, bool search_abs, bool full_ws,
    torch::Tensor bufs, torch::Tensor tranges,
    torch::Tensor n_tranges, torch::Tensor min_tranges){

    // # -- launch params --
    // w_threads = min(ws,32)
    // nthreads = (w_threads,w_threads)
    // ws_iters = (ws-1)//w_threads + 1
    // nblocks = (nq-1)//batches_per_block+1

   // fprintf(stdout,"qstart, nqueries: %d,%d\n",qstart,nqueries);
   // launch params 
   int ws_h_threads = std::min(ws_h,32);
   int ws_w_threads = std::min(ws_w,32);
   int ws_h_iters = ((ws_h-1)/ws_h_threads) + 1;
   int ws_w_iters = ((ws_w-1)/ws_w_threads) + 1;
   dim3 nthreads(ws_h_threads,ws_w_threads);

   int bpb = 2;
   int nblocks = ((nqueries - 1) / bpb) + 1;
   nblocks = min(nblocks,65535);
   bpb = ((nqueries - 1) / nblocks) + 1;

   // fprintf(stdout,"bpb,nblocks,w_threads: %d,%d,%d,%d\n",
   //         bpb,nblocks,ws_h_threads,ws_w_threads);
   // fprintf(stdout,"reflect_bounds,search_abs: %d,%d\n",reflect_bounds,search_abs);
    
   // launch kernel
   AT_DISPATCH_FLOATING_TYPES(vid0.type(), "dnls_search_forward_kernel", ([&] {
      l2_search_with_index_forward_kernel<scalar_t><<<nblocks, nthreads>>>(
        vid0.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        vid1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        fflow.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        bflow.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
        qstart, nqueries, stride0, n_h0, n_w0,
        h0_off, w0_off, h1_off, w1_off,
        ps, pt, ws_h, ws_w, wt, chnls, dilation, stride1,
        use_adj, reflect_bounds, search_abs, full_ws,
        bufs.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
        tranges.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
        n_tranges.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
        min_tranges.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
        ws_h_iters, ws_w_iters, bpb);
      }));
}


/****************************

       Backward Pass

****************************/

template <typename scalar_t>
__global__ void l2_search_with_index_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_vid0,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_vid1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid0,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid1,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> grad_dists,
    const torch::PackedTensorAccessor32<int,3,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> rand_nums,
    int qstart, int stride0, int n_h0, int n_w0,
    int h0_off, int w0_off, int h1_off, int w1_off,
    int ps, int pt, int dilation, bool use_adj, bool reflect_bounds,
    int bpt, int npt, int cpt) {

  // -- shape --
  int nq = grad_dists.size(0);
  int k =  grad_dists.size(1);
  int nframes = vid0.size(0);
  int colors = vid0.size(1);
  int height = vid0.size(2);
  int width = vid0.size(3);
  int n_hw0 = n_h0 * n_w0;

  // -- fwd decl registers --
  int ti,hi,wi;
  int tj,hj,wj;
  int tk,hk,wk;
  int tk_a,hk_a,wk_a;
  bool valid_hj,valid_wj;
  bool valid_hk,valid_wk;
  bool valid,valid_j,valid_k;
  float weight,pix,pix0,pix1;

  // -- declare constants --
  int psHalf = ps/2;
  int adj = use_adj ? psHalf : 0;

  // -- limits --
  int i0_max = inds.size(0);
  int i1_max = inds.size(1);

  // -- get indices --
  int i0_start = bpt * (threadIdx.x + blockDim.x * blockIdx.x);
  int i1_start = threadIdx.y * npt;
  int c0_start = threadIdx.z * cpt;

  // -- get block limits --
  int i0_end = min(i0_start + bpt,i0_max);
  int i1_end = min(i1_start + npt,i1_max);
  int c0_end = min(c0_start + cpt,colors);

  // -- color offset --
  int c0 = 0;
  int c0_dist = c0_end - c0_start;
  int c0_offset = 0;

  // -- each region --
  for (int i0=i0_start; i0 < i0_end; i0++){

    int qindex = i0 + qstart;
    int i_mod = qindex % n_hw0;
    tk_a = qindex / n_hw0;
    wk_a = ((i_mod % n_w0) * stride0) % width ;
    hk_a = ((i_mod / n_w0) * stride0) % height;
    c0_offset = __float2int_rd(c0_dist * rand_nums[i0][0][0]);

    // k neighbors
    for (int i1=i1_start; i1 < i1_end; i1++){
      ti = inds[i0][i1][0];
      hi = inds[i0][i1][1];
      wi = inds[i0][i1][2];
      weight = grad_dists[i0][i1];

      for (int pk = 0; pk < pt; pk++){
        for (int pi = 0; pi < ps; pi++){
          for (int pj = 0; pj < ps; pj++){
            

            // -- anchor patch --
            hk = (hk_a-h0_off) + dilation*(pi - psHalf + adj);
            hk = reflect_bounds ? bounds(hk,height) : hk;
            wk = (wk_a-w0_off) + dilation*(pj - psHalf + adj);
            wk = reflect_bounds ? bounds(wk,width) : wk;
            tk = reflect_bounds ? bounds(tk_a+pk,nframes) : tk_a+pk;

            // -- proposed location --
            hj = (hi-h1_off) + dilation*(pi - psHalf + adj);
            hj = reflect_bounds ? bounds(hj,height) : hj;
            wj = (wi-w1_off) + dilation*(pj - psHalf + adj);
            wj = reflect_bounds ? bounds(wj,width) : wj;
            tj = reflect_bounds ? bounds(ti+pk,nframes) : ti+pk;

            // -- assess if valid --
            valid_hj = (hj >= 0) && (hj < height);
            valid_wj = (wj >= 0) && (wj < width);
            valid_j = valid_hj && valid_wj;

            valid_hk = (hk >= 0) && (hk < height);
            valid_wk = (wk >= 0) && (wk < width);
            valid_k = valid_hk && valid_wk;

            for (int _c0 = c0_start; _c0 < c0_end; _c0++){
              c0 = (_c0 + c0_offset) % c0_dist + c0_start;
              pix0 =  valid_k ? vid0[tk][c0][hk][wk] : 0.;
              pix1 =  valid_j ? vid1[tj][c0][hj][wj] : 0.;
              pix = 2 * weight * (pix0 - pix1);

              if (valid_j){
                grad_vid1[tj][c0][hj][wj] -= pix;
              }
              if (valid_k){
                grad_vid0[tk][c0][hk][wk] += pix;
              }

            }
          }
        }
      }
    }
  }
}

void l2_search_with_index_backward_cuda(
    torch::Tensor grad_vid0, torch::Tensor grad_vid1,
    torch::Tensor vid0, torch::Tensor vid1,
    torch::Tensor grad_dists, torch::Tensor inds,
    int qstart, int stride0, int n_h0, int n_w0,
    int h0_off, int w0_off, int h1_off, int w1_off,
    int ps, int pt, int dilation,
    bool use_adj, bool reflect_bounds, bool exact) {

  // -- unpack --
  int nframes = vid0.size(0);
  int colors = vid0.size(1);
  int height = vid0.size(2);
  int width = vid0.size(3);
  int nqueries = inds.size(0);
  int k = grad_dists.size(1);
  assert(pt == 1);

  // -- compute number of neighbor threads --
  int npt = 8;
  int neigh_nthreads = (k-1) / npt + 1;
  if (neigh_nthreads > 64){
    neigh_nthreads = 64;
    npt = (k-1)/neigh_nthreads + 1;
  }
  if (exact){
    neigh_nthreads = 1;
    npt = k;
  }

  // -- compute number of color threads --
  int cpt = exact ? 1 : colors;
  int color_nthreads = (colors - 1)/cpt + 1;

  // -- compute number of blocks --
  //    [think: parallelization over "nqueries"]
  int bpt = 2;
  int query_nthreads = 16;
  int total_per_block = bpt * query_nthreads;
  int nblocks = ((nqueries - 1) / total_per_block) + 1;
  if (exact){
    bpt = nqueries;
    query_nthreads = 1;
    nblocks = 1;
  }

  // -- launch params --
  dim3 nthreads(query_nthreads, neigh_nthreads, color_nthreads);

  // -- info --
  // fprintf(stdout,
  //         "query_nthreads, neigh_nthreads, color_nthreads: %d,%d,%d\n",
  //         query_nthreads, neigh_nthreads, color_nthreads);
  // fprintf(stdout,"nblocks: %d\n",nblocks);
  // fprintf(stdout,"bpt,npt,cpt: %d,%d,%d\n",bpt,npt,cpt);
  // fprintf(stdout,"h0_off,w0_off,h1_off,w1_off: %d,%d,%d,%d\n",
  //         h0_off,w0_off,h1_off,w1_off);
  // fprintf(stdout,"ps,pt,dil: %d,%d,%d\n",ps,pt,dilation);

  // -- allocate random values --
  auto cu_index = grad_vid0.device().index();
  auto options = torch::TensorOptions().device(torch::kCUDA,
                                               cu_index).dtype(torch::kFloat32);
  torch::Tensor rand_nums = torch::rand({nqueries,1,1},options);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(vid0.type(), "dnls_search_backward_kernel", ([&] {
    l2_search_with_index_backward_kernel<scalar_t><<<nblocks, nthreads>>>(
        grad_vid0.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        grad_vid1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        vid0.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        vid1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        grad_dists.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        inds.packed_accessor32<int,3,torch::RestrictPtrTraits>(),
        rand_nums.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        qstart, stride0, n_h0, n_w0, h0_off, w0_off, h1_off, w1_off,
        ps,pt,dilation,use_adj,reflect_bounds,
        bpt,npt,cpt);
  }));

}


/****************************

       Remove Self

****************************/

__global__ void remove_self_from_search_kernel(
    const torch::PackedTensorAccessor32<int,3,torch::RestrictPtrTraits> inds,
    torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> mask,
    int qstart, int stride0, int n_h0, int n_w0, int qpb, int npt) {

  // -- shape --
  int nq = inds.size(0);
  int k =  inds.size(1);
  int n_hw0 = n_h0 * n_w0;

  // -- fwd decl registers --
  int ti,hi,wi;
  int tj,hj,wj;
  int qindex,i_mod;
  bool eq_ij;

  // -- boundary --
  int i0_max = inds.size(0);
  int i1_max = inds.size(1);

  // -- get indices --
  int i0_start = qpb * blockIdx.x;
  int i1_start = npt * threadIdx.x;

  // -- get block limits --
  int i0_end = min(i0_start + qpb,i0_max);
  int i1_end = min(i1_start + npt,i1_max);

  // -- each region --
  for (int i0=i0_start; i0 < i0_end; i0++){

    // -- index from i0 --
    qindex = i0 + qstart;
    i_mod = qindex % n_hw0;
    ti = qindex / n_hw0;
    wi = ((i_mod % n_w0) * stride0);
    hi = ((i_mod / n_w0) * stride0);

    // -- each neighbor --
    for (int i1=i1_start; i1 < i1_end; i1++){

      // -- neighbor index --
      tj = inds[i0][i1][0];
      hj = inds[i0][i1][1];
      wj = inds[i0][i1][2];

      // -- check valids --
      eq_ij = ti == tj;
      eq_ij = eq_ij && (hi == hj);
      eq_ij = eq_ij && (wi == wj);
      
      // -- assignment --
      mask[i0][i1] = eq_ij;
    }
  }

}

void remove_self_from_search_cuda(
    torch::Tensor inds, torch::Tensor mask,
    int qstart, int stride0, int n_h0, int n_w0) {

  // -- unpack --
  int nqueries = inds.size(0);
  int k = inds.size(1);
  int nneigh = k;

  // -- number of queries per cuda-block (qpb) --
  int qpb = 2;
  int query_nblocks = (nqueries-1)/qpb+1;
  query_nblocks = min(nqueries,512);
  qpb = (nqueries-1)/query_nblocks + 1;

  // -- compute number of neighbor per threads (npt) --
  int npt = 2;
  int neigh_nthreads = (nneigh-1)/npt+1;
  neigh_nthreads = min(nneigh,512);
  npt = (nneigh-1)/neigh_nthreads+1;
  // fprintf(stdout,"qpb,npt: %d,%d\n",qpb,npt);

  // -- launch kernel --
  remove_self_from_search_kernel<<<query_nblocks, neigh_nthreads>>>(
        inds.packed_accessor32<int,3,torch::RestrictPtrTraits>(),
        mask.packed_accessor32<bool,2,torch::RestrictPtrTraits>(),
        qstart, stride0, n_h0, n_w0, qpb, npt);
}

