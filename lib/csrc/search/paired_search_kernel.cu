#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstddef>
#include "paired_details.cu"

using namespace at;


/****************************

       Forward Pass

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> flow,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<int,6,torch::RestrictPtrTraits> inds,
    int ws_h, int ws_w, int ps,
    int stride0, int stride1, int dilation,
    int q_shift, int nH0, int nW0,
    bool reflect_bounds, bool full_ws, bool full_ws_time,
    int patch_offset, int off_H0, int off_W0, int off_H1, int off_W1,
    int q_per_thread, int ws_h_per_thread, int ws_w_per_thread){

  // -- unpack shape --
  int B = frame0.size(0);
  int HD = frame0.size(1);
  int C = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int Q = dists.size(2);

  // -- invalid constant --
  float invalid = __int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    invalid = -invalid;
  }

  // -- search region offsets --
  // int psHalf = (ps)/2;
  int wsHalf_h = (ws_h-1)/2;
  int wsHalf_w = (ws_w-1)/2;
  // int adj = use_adj ? psHalf : 0;
  int wsOff_h,wsOff_w;
  // int wsMax_h = stride1*(ws_h-1-wsHalf_h);
  // int wsMax_w = stride1*(ws_w-1-wsHalf_w);


  // -- cuda index --
  int ibatch = blockIdx.y;
  int ihead = blockIdx.z;
  int q_start = blockIdx.x*q_per_thread;
  int qi,ws_i,ws_j;

  // decls
  int ref_patch[2];
  int prop_patch[2];
  int frame_anchor[2];
  int ref_pix[2];
  int prop_pix[2];
  bool valid;
  bool valid_ref_patch,valid_prop_patch;
  bool valid_ref[3];
  bool valid_prop[3];

  // -- cleaner code --
  int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- indexing --
  int qindex,qindex_tmp;
  scalar_t dist,pix0,pix1,_dist;

  for (int q_index = 0; q_index < q_per_thread; q_index++){


    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }
    qindex = qi + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);
    check_bounds_2d(valid_ref_patch,ref_patch,H,W);

    // -- assign to reference --
    frame_anchor[0] = ref_patch[0];
    frame_anchor[1] = ref_patch[1];

    // -- compute frame offsets with flow --
    update_centers_flow(frame_anchor[0],frame_anchor[1],H,W,flow[ibatch]);

    // -- search region offsets --
    set_search_offsets(wsOff_h, wsOff_w, frame_anchor[0], frame_anchor[1], stride1,
                       wsHalf_h, wsHalf_w, ws_h, ws_w, H, W, full_ws_time);

    // ---------------------------------------
    //          spatial searching
    // ---------------------------------------

    // -- search across space --
    for (int _xi = 0; _xi < ws_h_per_thread; _xi++){
      ws_i = threadIdx.x + blockDim.x*_xi;
      if (ws_i >= ws_h){ continue; }
      for (int _yi = 0; _yi < ws_w_per_thread; _yi++){
        ws_j = threadIdx.y + blockDim.y*_yi;
        if (ws_j >= ws_w){ continue; }

        // -- compute proposed location --
        prop_patch[0] = frame_anchor[0] + stride1 * (ws_i - wsOff_h);
        prop_patch[1] = frame_anchor[1] + stride1 * (ws_j - wsOff_w);
        check_bounds_2d(valid_prop_patch,prop_patch,H,W);
        valid = valid_ref_patch && valid_prop_patch;

        // -- init dist --
        dist = 0;

        //  -- compute patch difference --
        if (valid){

          compute_dist_2d<scalar_t,DIST_TYPE>(dist,
                       frame0[ibatch][ihead],frame1[ibatch][ihead],
                       ref_patch, prop_patch, 
                       ref_pix, prop_pix, valid_ref, valid_prop,
                       ps,dilation,reflect_bounds,
                       patch_offset,center_offsets,invalid,
                       C,H,W,pix0,pix1,_dist);

        }

        // -- assignent --
        if (!valid){ dist = invalid; }
        dists[ibatch][ihead][qi][ws_i][ws_j] = dist;
        inds[ibatch][ihead][qi][ws_i][ws_j][0] = prop_patch[0];
        inds[ibatch][ihead][qi][ws_i][ws_j][1] = prop_patch[1];
          
      }
    }
  }
}

void paired_search_forward_cuda(
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow, torch::Tensor dists, torch::Tensor inds,
    int ps, int k, int dist_type,
    int stride0, int stride1, int dilation, int q_shift,
    bool reflect_bounds, bool full_ws, bool full_ws_time,
    bool use_adj, int off_H0, int off_W0, int off_H1, int off_W1){

   // -- derived quantities --
   int B = frame0.size(0);
   int HD = frame0.size(1);
   int H = frame0.size(3);
   int W = frame0.size(4);
   int nH0 = (H-1)/stride0+1;
   int nW0 = (W-1)/stride0+1;

   // -- threads --
   int nheads = dists.size(1);
   int nqueries = dists.size(2);
   int ws_h = dists.size(3);
   int ws_w = dists.size(4);
   int ws_h_threads = std::min(ws_h,27);
   int ws_w_threads = std::min(ws_w,27);
   int ws_h_per_thread = ((ws_h-1)/ws_h_threads) + 1;
   int ws_w_per_thread = ((ws_w-1)/ws_w_threads) + 1;
   dim3 nthreads(ws_h_threads,ws_w_threads);

   // -- nblocks --
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   dim3 nblocks(nquery_blocks,B,HD);

   // -- share --
   int psHalf = ps/2;
   int adj = use_adj ? psHalf : 0;
   // int patch_offset = adj - psHalf;
   int patch_offset = adj - psHalf;

   // -- viz --
   // fprintf(stdout,"ws_h,ws_w: %d,%d\n",ws_h,ws_w);
   // fprintf(stdout,"nquery_blocks,B,HD: %d,%d,%d\n",nquery_blocks,B,HD);

   // launch kernel
   if (dist_type == 0){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_forward_kernel", ([&] {
       paired_search_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
            ws_h, ws_w, ps, stride0, stride1, dilation, 
            q_shift, nH0, nW0, reflect_bounds, full_ws, full_ws_time,
            patch_offset, off_H0, off_W0, off_H1, off_W1,
            q_per_thread, ws_h_per_thread, ws_w_per_thread);
          }));
   }else if(dist_type == 1){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_forward_kernel", ([&] {
       paired_search_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
            ws_h, ws_w, ps, stride0, stride1, dilation, 
            q_shift, nH0, nW0, reflect_bounds, full_ws, full_ws_time,
            patch_offset, off_H0, off_W0, off_H1, off_W1,
            q_per_thread, ws_h_per_thread, ws_w_per_thread);
          }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}


/**********************************

      Forward Pass (Bilin2d)

**********************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_bilin2d_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> flow,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> inds,
    int ws_h, int ws_w, int ps,
    int stride0, float _stride1, int dilation,
    int q_shift, int nH0, int nW0,
    bool reflect_bounds, bool full_ws, bool full_ws_time,
    int patch_offset, int off_H0, int off_W0, int off_H1, int off_W1,
    int q_per_thread, int ws_h_per_thread, int ws_w_per_thread){

  // -- unpack shape --
  int B = frame0.size(0);
  int HD = frame0.size(1);
  int C = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int Q = dists.size(2);
  scalar_t stride1 = static_cast<scalar_t>(_stride1);

  // -- invalid constant --
  float _invalid = __int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    _invalid = -_invalid;
  }
  scalar_t invalid = static_cast<scalar_t>(_invalid);

  // -- search region offsets --
  // int psHalf = (ps)/2;
  // int wsHalf_h = (ws_h)/2;
  // int wsHalf_w = (ws_w)/2;
  // int wsMax_h = stride1*(ws_h-1-wsHalf_h);
  // int wsMax_w = stride1*(ws_w-1-wsHalf_w);
  // int adj = use_adj ? psHalf : 0;

  // int wsHalf_h = (ws_h-1)/2;
  // int wsHalf_w = (ws_w-1)/2;
  // int wsOff_h,wsOff_w;

  scalar_t wsHalf_h = trunc((ws_h)/2);
  scalar_t wsHalf_w = trunc((ws_w)/2);
  scalar_t wsOff_h,wsOff_w;

  // -- cuda index --
  int ibatch = blockIdx.y;
  int ihead = blockIdx.z;
  int q_start = blockIdx.x*q_per_thread;
  int qi,ws_i,ws_j;

  // decls
  int ref_patch[2];
  scalar_t prop_patch[2];
  scalar_t frame_anchor[2];
  int ref_pix[2];
  scalar_t prop_pix[2];
  int prop_i[2];
  bool valid;
  bool valid_ref_patch,valid_prop_patch;
  bool valid_ref[3];
  bool valid_prop[3];

  // -- cleaner code --
  int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- indexing --
  int qindex,qindex_tmp;
  scalar_t dist,pix0,pix1,_dist;

  for (int q_index = 0; q_index < q_per_thread; q_index++){


    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }
    qindex = qi + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);
    check_bounds_2d(valid_ref_patch,ref_patch,H,W);

    // -- assign to reference --
    frame_anchor[0] = __int2float_rn(ref_patch[0]);
    frame_anchor[1] = __int2float_rn(ref_patch[1]);

    // -- compute frame offsets with flow --
    update_centers_flow(frame_anchor[0],frame_anchor[1],H,W,flow[ibatch]);


    // -- search region offsets --
    set_search_offsets<scalar_t>(wsOff_h, wsOff_w,
                                 frame_anchor[0], frame_anchor[1],
                                 stride1, wsHalf_h, wsHalf_w,
                                 ws_h, ws_w, H, W, full_ws_time);

    // ---------------------------------------
    //          spatial searching
    // ---------------------------------------

    // -- search across space --
    for (int _xi = 0; _xi < ws_h_per_thread; _xi++){
      ws_i = threadIdx.x + blockDim.x*_xi;
      if (ws_i >= ws_h){ continue; }
      for (int _yi = 0; _yi < ws_w_per_thread; _yi++){
        ws_j = threadIdx.y + blockDim.y*_yi;
        if (ws_j >= ws_w){ continue; }

        // -- compute proposed location --
        prop_patch[0] = frame_anchor[0] + stride1 * (ws_i - wsOff_h);
        prop_patch[1] = frame_anchor[1] + stride1 * (ws_j - wsOff_w);
        check_bounds_2d(valid_prop_patch,prop_patch,H,W);
        valid = valid_ref_patch && valid_prop_patch;

        // -- init dist --
        dist = 0;

        //  -- compute patch difference --
        if (valid){
          compute_dist_bilin2d_2d<scalar_t,DIST_TYPE>(dist,
                       frame0[ibatch][ihead],
                       frame1[ibatch][ihead],
                       ref_patch, prop_patch,
                       ref_pix, prop_pix, prop_i,
                       valid_ref, valid_prop,
                       ps,dilation,reflect_bounds,
                       patch_offset,center_offsets,invalid,
                       C,H,W,pix0,pix1,_dist);
        }

        // -- assignent --
        if (!valid){ dist = invalid; }
        dists[ibatch][ihead][qi][ws_i][ws_j] = dist;
        inds[ibatch][ihead][qi][ws_i][ws_j][0] = prop_patch[0];
        inds[ibatch][ihead][qi][ws_i][ws_j][1] = prop_patch[1];
          
      }
    }
  }
}

void paired_search_bilin2d_forward_cuda(
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow, torch::Tensor dists, torch::Tensor inds,
    int ps, int k, int dist_type,
    int stride0, float stride1, int dilation, int q_shift,
    bool reflect_bounds, bool full_ws, bool full_ws_time,
    bool use_adj, int off_H0, int off_W0, int off_H1, int off_W1){

   // -- derived quantities --
   int B = frame0.size(0);
   int HD = frame0.size(1);
   int H = frame0.size(3);
   int W = frame0.size(4);
   int nH0 = (H-1)/stride0+1;
   int nW0 = (W-1)/stride0+1;

   // -- threads --
   int nheads = dists.size(1);
   int nqueries = dists.size(2);
   int ws_h = dists.size(3);
   int ws_w = dists.size(4);
   int ws_h_threads = std::min(ws_h,27);
   int ws_w_threads = std::min(ws_w,27);
   int ws_h_per_thread = ((ws_h-1)/ws_h_threads) + 1;
   int ws_w_per_thread = ((ws_w-1)/ws_w_threads) + 1;
   dim3 nthreads(ws_h_threads,ws_w_threads);

   // -- nblocks --
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   dim3 nblocks(nquery_blocks,B,HD);

   // -- share --
   int psHalf = ps/2;
   int adj = use_adj ? psHalf : 0;
   // int patch_offset = adj - psHalf;
   int patch_offset = adj - psHalf;

   // -- viz --
   // fprintf(stdout,"ws_h,ws_w: %d,%d\n",ws_h,ws_w);
   // fprintf(stdout,"nquery_blocks,B,HD: %d,%d,%d\n",nquery_blocks,B,HD);

   // launch kernel
   if (dist_type == 0){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                                  "paired_search_bilin2d_forward_kernel", ([&] {
       paired_search_bilin2d_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            ws_h, ws_w, ps, stride0, stride1, dilation, 
            q_shift, nH0, nW0, reflect_bounds, full_ws, full_ws_time,
            patch_offset, off_H0, off_W0, off_H1, off_W1,
            q_per_thread, ws_h_per_thread, ws_w_per_thread);
          }));
   }else if(dist_type == 1){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                                  "paired_search_bilin2d_forward_kernel", ([&] {
       paired_search_bilin2d_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            ws_h, ws_w, ps, stride0, stride1, dilation, 
            q_shift, nH0, nW0, reflect_bounds, full_ws, full_ws_time,
            patch_offset, off_H0, off_W0, off_H1, off_W1,
            q_per_thread, ws_h_per_thread, ws_w_per_thread);
          }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}


/****************************

       Backward Pass

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame0,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_dists,
    const torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> inds,
    int q_shift, int stride0, int nH0, int nW0,
    int off_H0, int off_W0, int off_H1, int off_W1,
    int ps, int pt, int dilation, int patch_offset,
    bool reflect_bounds, int ftrs_per_thread) {

  // -- shape --
  int nbatch = grad_dists.size(0);
  int Q = grad_dists.size(2);
  int K =  grad_dists.size(3);
  int HD = frame0.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);

  // -- fwd decl registers --
  int ref_patch[2];
  int prop_patch[2];
  int ref[2];
  int prop[2];
  bool valid_ref[3];
  bool valid_prop[3];
  int qindex,qindex_tmp;
  bool valid;
  scalar_t weight,pix0,pix1,pix;
  int iftr;
  int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- location to fill --
  int i0 = blockIdx.x*blockDim.x+threadIdx.x;
  int i1 = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/nbatch;
  int ibatch = (blockIdx.z-ihead*nbatch) % nbatch;

  // -- feature chunk --
  int ftr_start = threadIdx.z * ftrs_per_thread;
  int ftr_end = min(F,ftr_start + ftrs_per_thread);

  // -- each region --
  if ((i0 < Q) && (i1 < K)){

    // -- full-resolution video query index --
    qindex = i0 + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);

    // -- proposed location --
    prop_patch[0] = inds[ibatch][ihead][i0][i1][0];
    prop_patch[1] = inds[ibatch][ihead][i0][i1][1];
    prop_patch[2] = inds[ibatch][ihead][i0][i1][2];
    weight = grad_dists[ibatch][ihead][i0][i1];

    // -- update patch --
    update_bwd_patch_2d<scalar_t,DIST_TYPE>(
                     grad_frame0[ibatch][ihead],grad_frame1[ibatch][ihead],
                     frame0[ibatch][ihead],frame1[ibatch][ihead],
                     weight,ref_patch,prop_patch,
                     ps,pt,dilation,reflect_bounds,
                     center_offsets,patch_offset,
                     iftr,ftr_start,ftr_end,
                     ref,prop,valid_ref,valid_prop,valid,
                     H,W,pix0,pix1,pix,i1);

  }
}

void paired_search_backward_cuda(
    torch::Tensor grad_frame0, torch::Tensor grad_frame1,
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor grad_dists, const torch::Tensor inds,
    int q_shift, int stride0, int nH0, int nW0,
    int ps, int pt, int dilation, bool reflect_bounds, bool use_adj,
    int off_H0, int off_W0, int off_H1, int off_W1, int dist_type) {

  // -- unpack --
  int B = frame0.size(0);
  int HD = frame0.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int nqueries = inds.size(2);
  int K = inds.size(3);
  int BHD = B*HD;
  assert(pt == 1);

  // -- launch parameters --
  int nbatch = grad_dists.size(0);
  int nheads = grad_dists.size(1);
  int nq = grad_dists.size(2);
  int k = grad_dists.size(3);
  int ftr_threads = min(15,F);
  dim3 threadsPerBlock(10,4,ftr_threads);
  dim3 blocksPerGrid(1, 1, nheads*nbatch);
  blocksPerGrid.x = ceil(double(nq)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(k)/double(threadsPerBlock.y));
  int ftrs_per_thread = (F-1)/ftr_threads+1;

  // -- shared --
  int psHalf = ps/2;
  int adj = use_adj ? psHalf : 0;
  int patch_offset = adj - psHalf;
  // int patch_offset = psHalf - adj;

  // -- launch kernel --
  if (dist_type == 0){ // prod
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_backward_kernel", ([&] {
    paired_search_backward_kernel<scalar_t,0><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          q_shift, stride0, nH0, nW0, off_H0, off_W0, off_H1, off_W1,
          ps, pt, dilation, patch_offset, reflect_bounds, ftrs_per_thread);
    }));
  }else if (dist_type == 1){ // l2
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_backward_kernel", ([&] {
    paired_search_backward_kernel<scalar_t,1><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          q_shift, stride0, nH0, nW0, off_H0, off_W0, off_H1, off_W1,
          ps, pt, dilation, patch_offset, reflect_bounds, ftrs_per_thread);
    }));
  }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");    }


}



/****************************

       Backward Bilinear-2d

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_bilin2d_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame0,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_dists,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds,
    int q_shift, int stride0, int nH0, int nW0,
    int off_H0, int off_W0, int off_H1, int off_W1,
    int ps, int pt, int dilation, int patch_offset,
    bool reflect_bounds, int ftrs_per_thread) {

  // -- shape --
  int nbatch = grad_dists.size(0);
  int Q = grad_dists.size(2);
  int K =  grad_dists.size(3);
  int HD = frame0.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);

  // -- fwd decl registers --
  int ref_patch[2];
  int prop_patch[2];
  int ref[2];
  int prop[2];
  bool valid_ref[3];
  bool valid_prop[3];
  int qindex,qindex_tmp;
  bool valid;
  scalar_t weight,pix0,pix1,pix;
  int iftr;
  int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- location to fill --
  int i0 = blockIdx.x*blockDim.x+threadIdx.x;
  int i1 = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/nbatch;
  int ibatch = (blockIdx.z-ihead*nbatch) % nbatch;

  // -- feature chunk --
  int ftr_start = threadIdx.z * ftrs_per_thread;
  int ftr_end = min(F,ftr_start + ftrs_per_thread);

  // -- each region --
  if ((i0 < Q) && (i1 < K)){

    // -- full-resolution video query index --
    qindex = i0 + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);

    // -- proposed location --
    prop_patch[0] = inds[ibatch][ihead][i0][i1][0];
    prop_patch[1] = inds[ibatch][ihead][i0][i1][1];
    prop_patch[2] = inds[ibatch][ihead][i0][i1][2];
    weight = grad_dists[ibatch][ihead][i0][i1];

    // -- update patch --
    update_bwd_patch_2d<scalar_t,DIST_TYPE>(
                     grad_frame0[ibatch][ihead],grad_frame1[ibatch][ihead],
                     frame0[ibatch][ihead],frame1[ibatch][ihead],
                     weight,ref_patch,prop_patch,
                     ps,pt,dilation,reflect_bounds,
                     center_offsets,patch_offset,
                     iftr,ftr_start,ftr_end,
                     ref,prop,valid_ref,valid_prop,valid,
                     H,W,pix0,pix1,pix,i1);

  }
}

void paired_search_bilin2d_backward_cuda(
    torch::Tensor grad_frame0, torch::Tensor grad_frame1,
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor grad_dists, const torch::Tensor inds,
    int q_shift, int stride0, int nH0, int nW0,
    int ps, int pt, int dilation, bool reflect_bounds, bool use_adj,
    int off_H0, int off_W0, int off_H1, int off_W1, int dist_type) {

  // -- unpack --
  int B = frame0.size(0);
  int HD = frame0.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int nqueries = inds.size(2);
  int K = inds.size(3);
  int BHD = B*HD;
  assert(pt == 1);

  // -- launch parameters --
  int nbatch = grad_dists.size(0);
  int nheads = grad_dists.size(1);
  int nq = grad_dists.size(2);
  int k = grad_dists.size(3);
  int ftr_threads = min(15,F);
  dim3 threadsPerBlock(10,4,ftr_threads);
  dim3 blocksPerGrid(1, 1, nheads*nbatch);
  blocksPerGrid.x = ceil(double(nq)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(k)/double(threadsPerBlock.y));
  int ftrs_per_thread = (F-1)/ftr_threads+1;

  // -- shared --
  int psHalf = ps/2;
  int adj = use_adj ? psHalf : 0;
  int patch_offset = adj - psHalf;
  // int patch_offset = psHalf - adj;

  // -- launch kernel --
  if (dist_type == 0){ // prod
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                               "paired_search_bilin2d_backward_kernel", ([&] {
    paired_search_bilin2d_backward_kernel<scalar_t,0><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          q_shift, stride0, nH0, nW0, off_H0, off_W0, off_H1, off_W1,
          ps, pt, dilation, patch_offset, reflect_bounds, ftrs_per_thread);
    }));
  }else if (dist_type == 1){ // l2
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                               "paired_search_bilin2d_backward_kernel", ([&] {
    paired_search_bilin2d_backward_kernel<scalar_t,1><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          q_shift, stride0, nH0, nW0, off_H0, off_W0, off_H1, off_W1,
          ps, pt, dilation, patch_offset, reflect_bounds, ftrs_per_thread);
    }));
  }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");    }


}


