#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstddef>
#include "paired_details.cu"

using namespace at;


/****************************

       Forward Pass

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_int_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> flow,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<int,6,torch::RestrictPtrTraits> inds,
    int ws, int ps, int stride0, int stride1, int dilation,
    bool reflect_bounds, bool full_ws, int patch_offset,
    int nW0, int q_per_thread, int ws_per_thread){

  // -- unpack shape --
  int B = frame0.size(0);
  int HD_frame = frame0.size(1);
  int HD_flow = flow.size(1);
  int C = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int Q = dists.size(2);
  int HD = max(HD_frame,HD_flow);

  // -- invalid constant --
  float invalid = __int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    invalid = -invalid;
  }


  // -- search region offsets --
  // int psHalf = (ps)/2;
  int wsHalf = (ws-1)/2;
  // int wsHalf_w = (ws_w)/2;
  // int adj = use_adj ? psHalf : 0;
  int wsOff_h,wsOff_w;
  // int wsMax_h = stride1*(ws_h-1-wsHalf_h);
  // int wsMax_w = stride1*(ws_w-1-wsHalf_w);

  // -- cuda index --
  int ibatch = blockIdx.y;
  int ihead = blockIdx.z;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int q_start = blockIdx.x*q_per_thread;
  int qi,ws_i,ws_j;

  // decls
  int ref_patch[2];
  int prop_patch[2];
  int frame_anchor[2];
  int ref_pix[2];
  int prop_pix[2];
  bool valid;
  bool valid_ref_patch,valid_prop_patch;
  bool valid_ref[3];
  bool valid_prop[3];

  // -- cleaner code --
  // int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- indexing --
  int qindex,qindex_tmp;
  scalar_t dist,pix0,pix1,_dist;

  for (int q_index = 0; q_index < q_per_thread; q_index++){


    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }
    qindex = qi;// + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);
    check_bounds_2d(valid_ref_patch,ref_patch,H,W);

    // -- assign to reference --
    frame_anchor[0] = ref_patch[0];
    frame_anchor[1] = ref_patch[1];

    // -- compute frame offsets with flow --
    update_centers_flow(frame_anchor[0],frame_anchor[1],H,W,flow[ibatch][ihead_fl]);

    // -- search region offsets --
    set_search_offsets(wsOff_h, wsOff_w,
                       frame_anchor[0], frame_anchor[1], stride1,
                       wsHalf, ws, H, W, full_ws);

    // ---------------------------------------
    //          spatial searching
    // ---------------------------------------

    // -- search across space --
    for (int _xi = 0; _xi < ws_per_thread; _xi++){
      ws_i = threadIdx.x + blockDim.x*_xi;
      if (ws_i >= ws){ continue; }
      for (int _yi = 0; _yi < ws_per_thread; _yi++){
        ws_j = threadIdx.y + blockDim.y*_yi;
        if (ws_j >= ws){ continue; }

        // -- compute proposed location --
        prop_patch[0] = frame_anchor[0] + stride1 * (ws_i - wsOff_h);
        prop_patch[1] = frame_anchor[1] + stride1 * (ws_j - wsOff_w);
        check_bounds_2d(valid_prop_patch,prop_patch,H,W);
        valid = valid_ref_patch && valid_prop_patch;

        // -- init dist --
        dist = 0;

        //  -- compute patch difference --
        if (valid){

          compute_dist_2d<scalar_t,DIST_TYPE>(dist,
                       frame0[ibatch][ihead_fr],frame1[ibatch][ihead_fr],
                       ref_patch, prop_patch, 
                       ref_pix, prop_pix, valid_ref, valid_prop,
                       ps,dilation,reflect_bounds,
                       patch_offset,invalid,
                       C,H,W,pix0,pix1,_dist);

        }

        // -- assignent --
        if (!valid){ dist = invalid; }
        dists[ibatch][ihead_fl][qi][ws_i][ws_j] = dist;
        inds[ibatch][ihead_fl][qi][ws_i][ws_j][0] = prop_patch[0];
        inds[ibatch][ihead_fl][qi][ws_i][ws_j][1] = prop_patch[1];
          
      }
    }
  }
}

void paired_search_int_forward_cuda(
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow, torch::Tensor dists, torch::Tensor inds,
    int ps, int k, int stride0, int stride1, int dilation,
    bool reflect_bounds, bool full_ws, int patch_offset, int dist_type){

   // -- derived quantities --
   int B = frame0.size(0);
   int HD_frame = frame0.size(1);
   int HD_flow = flow.size(1);
   int H = frame0.size(3);
   int W = frame0.size(4);
   // int nH0 = (H-1)/stride0+1;
   int nW0 = (W-1)/stride0+1;
   int HD = max(HD_frame,HD_flow);

   // -- threads --
   int nqueries = dists.size(2);
   int ws = dists.size(3);
   int ws_threads = std::min(ws,25);
   int ws_per_thread = ((ws-1)/ws_threads) + 1;
   dim3 nthreads(ws_threads,ws_threads);

   // -- nblocks --
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   dim3 nblocks(nquery_blocks,B,HD);

   // -- share --
   // int psHalf = ps/2;
   // int adj = use_adj ? psHalf : 0;
   // // int patch_offset = adj - psHalf;
   // int patch_offset = adj - psHalf;

   // -- viz --
   // fprintf(stdout,"ws_h,ws_w: %d,%d,%d,%d\n",ws_h,ws_w,ws_h_threads,ws_h_per_thread);
   // fprintf(stdout,"nquery_blocks,B,HD: %d,%d,%d\n",nquery_blocks,B,HD);


   // launch kernel
   if (dist_type == 0){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_int_forward_kernel", ([&] {
       paired_search_int_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, nW0, q_per_thread, ws_per_thread);
          }));
   }else if(dist_type == 1){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_int_forward_kernel", ([&] {
       paired_search_int_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, nW0, q_per_thread, ws_per_thread);
          }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}


/**********************************

      Forward Pass (Bilin2d)

**********************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_bilin2d_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> flow,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> inds,
    int ws, int ps, int stride0, float _stride1, int dilation,
    bool reflect_bounds, bool full_ws, int patch_offset,
    int nW0, int q_per_thread, int ws_per_thread){

  // -- unpack shape --
  int B = frame0.size(0);
  int HD_frame = frame0.size(1);
  int HD_flow = flow.size(1);
  int HD_search = inds.size(1);
  int C = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int Q = dists.size(2);
  int HD = max(HD_frame,HD_flow);
  scalar_t stride1 = static_cast<scalar_t>(_stride1);

  // -- invalid constant --
  float _invalid = __int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    _invalid = -_invalid;
  }
  scalar_t invalid = static_cast<scalar_t>(_invalid);

  // -- search region offsets --
  // int psHalf = (ps)/2;
  // int wsHalf_h = (ws_h)/2;
  // int wsHalf_w = (ws_w)/2;
  // int wsMax_h = stride1*(ws_h-1-wsHalf_h);
  // int wsMax_w = stride1*(ws_w-1-wsHalf_w);
  // int adj = use_adj ? psHalf : 0;

  // int wsHalf_h = (ws_h-1)/2;
  // int wsHalf_w = (ws_w-1)/2;
  // int wsOff_h,wsOff_w;
  scalar_t wsHalf = trunc((ws-1)/2);
  scalar_t wsOff_h,wsOff_w;

  // -- cuda index --
  int ibatch = blockIdx.y;
  int ihead = blockIdx.z;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int ihead_sr = ihead % HD_search;
  int q_start = blockIdx.x*q_per_thread;
  int qi,ws_i,ws_j;

  // decls
  int ref_patch[2];
  scalar_t prop_patch[2];
  scalar_t frame_anchor[2];
  int ref_pix[2];
  scalar_t prop_pix[2];
  int prop_i[2];
  bool valid;
  bool valid_ref_patch,valid_prop_patch;
  bool valid_ref[3];
  bool valid_prop[3];

  // -- cleaner code --
  // int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- indexing --
  int qindex,qindex_tmp;
  scalar_t dist,pix0,pix1,_dist;

  for (int q_index = 0; q_index < q_per_thread; q_index++){


    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }
    qindex = qi;// + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);
    check_bounds_2d(valid_ref_patch,ref_patch,H,W);

    // -- assign to reference --
    frame_anchor[0] = __int2float_rn(ref_patch[0]);
    frame_anchor[1] = __int2float_rn(ref_patch[1]);

    // -- compute frame offsets with flow --
    // update_centers_flow(frame_anchor[0],frame_anchor[1],H,W,flow[ibatch][ihead_fl]);
    frame_anchor[0] += flow[ibatch][ihead_fl][1][ref_patch[0]][ref_patch[1]];
    frame_anchor[1] += flow[ibatch][ihead_fl][0][ref_patch[0]][ref_patch[1]];
    frame_anchor[0] = bounds(frame_anchor[0],H);
    frame_anchor[1] = bounds(frame_anchor[1],W);
    // frame_anchor[0] = floorf(frame_anchor[0]*1000)/(float)1000;
    // frame_anchor[1] = floorf(frame_anchor[1]*1000)/(float)1000;

    // -- search region offsets --
    set_search_offsets(wsOff_h, wsOff_w,
                       frame_anchor[0], frame_anchor[1], stride1,
                       wsHalf, ws, H, W, full_ws);

    // ---------------------------------------
    //          spatial searching
    // ---------------------------------------

    // -- search across space --
    for (int _xi = 0; _xi < ws_per_thread; _xi++){
      ws_i = threadIdx.x + blockDim.x*_xi;
      if (ws_i >= ws){ continue; }
      for (int _yi = 0; _yi < ws_per_thread; _yi++){
        ws_j = threadIdx.y + blockDim.y*_yi;
        if (ws_j >= ws){ continue; }


        // -- compute proposed location --
        prop_patch[0] = frame_anchor[0] + stride1 * (ws_i - wsOff_h);
        prop_patch[1] = frame_anchor[1] + stride1 * (ws_j - wsOff_w);
        check_bounds_2d<scalar_t>(valid_prop_patch,prop_patch,H,W);
        valid = valid_ref_patch && valid_prop_patch;


        // -- init dist --
        dist = 0;

        //  -- compute patch difference --
        if (valid){
          compute_dist_bilin2d_2d<scalar_t,DIST_TYPE>(dist,
                       frame0[ibatch][ihead_fr],
                       frame1[ibatch][ihead_fr],
                       ref_patch, prop_patch,
                       ref_pix, prop_pix, prop_i,
                       valid_ref, valid_prop,
                       ps,dilation,reflect_bounds,
                       patch_offset,invalid,
                       C,H,W,pix0,pix1,_dist);
        }


        // -- assignent --
        if (!valid){ dist = invalid; }
        dists[ibatch][ihead_sr][qi][ws_i][ws_j] = dist;
        inds[ibatch][ihead_sr][qi][ws_i][ws_j][0] = prop_patch[0];
        inds[ibatch][ihead_sr][qi][ws_i][ws_j][1] = prop_patch[1];
        // inds[ibatch][ihead_fl][qi][ws_i][ws_j][0] = frame_anchor[0];
        // inds[ibatch][ihead_fl][qi][ws_i][ws_j][1] = frame_anchor[1];
          
      }
    }
  }
}

void paired_search_bilin2d_forward_cuda(
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow, torch::Tensor dists, torch::Tensor inds,
    int ps, int k, int stride0, float stride1, int dilation,
    bool reflect_bounds, bool full_ws, int patch_offset, int dist_type){

   // -- derived quantities --
   int B = frame0.size(0);
   int HD_frame = frame0.size(1);
   int HD_flow = flow.size(1);
   int H = frame0.size(3);
   int W = frame0.size(4);
   // int nH0 = (H-1)/stride0+1;
   int nW0 = (W-1)/stride0+1;
   int HD = max(HD_frame,HD_flow);

   // -- threads --
   int nqueries = dists.size(2);
   int ws = dists.size(3);
   int ws_threads = std::min(ws,25);
   int ws_per_thread = ((ws-1)/ws_threads) + 1;
   dim3 nthreads(ws_threads,ws_threads);

   // -- nblocks --
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   dim3 nblocks(nquery_blocks,B,HD);

   // -- share --
   // int psHalf = ps/2;
   // int adj = use_adj ? psHalf : 0;
   // // int patch_offset = adj - psHalf;
   // int patch_offset = adj - psHalf;

   // -- viz --
   // fprintf(stdout,"ws_h,ws_w: %d,%d\n",ws_h,ws_w);
   // fprintf(stdout,"nquery_blocks,B,HD: %d,%d,%d\n",nquery_blocks,B,HD);

   // launch kernel
   if (dist_type == 0){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                                  "paired_search_bilin2d_forward_kernel", ([&] {
       paired_search_bilin2d_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, nW0, q_per_thread, ws_per_thread);
          }));
   }else if(dist_type == 1){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                                  "paired_search_bilin2d_forward_kernel", ([&] {
       paired_search_bilin2d_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, nW0, q_per_thread, ws_per_thread);
          }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}


/****************************

       Backward Pass

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame0,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_dists,
    const torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> inds,
    int stride0, int ps, int pt, int dilation, int patch_offset,
    bool reflect_bounds, int nW0, int ftrs_per_thread) {

  // -- shape --
  int nbatch = grad_dists.size(0);
  int Q = grad_dists.size(2);
  int K =  grad_dists.size(3);
  int HD_frame = frame0.size(1);
  int HD_flow = grad_dists.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int HD = max(HD_frame,HD_flow);

  // -- fwd decl registers --
  int ref_patch[2];
  int prop_patch[2];
  int ref[2];
  int prop[2];
  bool valid_ref[3];
  bool valid_prop[3];
  int qindex,qindex_tmp;
  bool valid;
  scalar_t weight,pix0,pix1,pix;
  int iftr;
  // int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- location to fill --
  int i0 = blockIdx.x*blockDim.x+threadIdx.x;
  int i1 = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/nbatch;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int ibatch = (blockIdx.z-ihead*nbatch) % nbatch;

  // -- feature chunk --
  int ftr_start = threadIdx.z * ftrs_per_thread;
  int ftr_end = min(F,ftr_start + ftrs_per_thread);

  // -- each region --
  if ((i0 < Q) && (i1 < K)){

    // -- full-resolution video query index --
    qindex = i0;// + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);

    // -- proposed location --
    prop_patch[0] = inds[ibatch][ihead_fl][i0][i1][0];
    prop_patch[1] = inds[ibatch][ihead_fl][i0][i1][1];
    prop_patch[2] = inds[ibatch][ihead_fl][i0][i1][2];
    weight = grad_dists[ibatch][ihead_fl][i0][i1];

    // -- update patch --
    update_bwd_patch_2d<scalar_t,DIST_TYPE>(
                     grad_frame0[ibatch][ihead_fr],
                     grad_frame1[ibatch][ihead_fr],
                     frame0[ibatch][ihead_fr],
                     frame1[ibatch][ihead_fr],
                     weight,ref_patch,prop_patch,
                     ps,pt,dilation,reflect_bounds,
                     patch_offset,iftr,ftr_start,ftr_end,
                     ref,prop,valid_ref,valid_prop,valid,
                     H,W,pix0,pix1,pix,i1);

  }
}

void paired_search_backward_cuda(
    torch::Tensor grad_frame0, torch::Tensor grad_frame1,
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor grad_dists, const torch::Tensor inds,
    int stride0, int ps, int pt, int dilation, bool reflect_bounds,
    int patch_offset, int dist_type) {


  // -- unpack --
  int B = frame0.size(0);
  int HD_frame = frame0.size(1);
  int HD_flow = grad_dists.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int HD = max(HD_frame,HD_flow);
  int nqueries = inds.size(2);
  int K = inds.size(3);
  int BHD = B*HD;
  assert(pt == 1);

  // -- launch parameters --
  int nbatch = grad_dists.size(0);
  int nq = grad_dists.size(2);
  int k = grad_dists.size(3);
  int ftr_threads = min(15,F);
  dim3 threadsPerBlock(10,4,ftr_threads);
  dim3 blocksPerGrid(1, 1, nbatch*HD);
  blocksPerGrid.x = ceil(double(nq)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(k)/double(threadsPerBlock.y));
  int ftrs_per_thread = (F-1)/ftr_threads+1;

  // -- shared --
  int nW0 = (W-1)/stride0+1;
  // int psHalf = ps/2;
  // int adj = use_adj ? psHalf : 0;
  // int patch_offset = adj - psHalf;
  // int patch_offset = psHalf - adj;

  // -- launch kernel --
  if (dist_type == 0){ // prod
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_backward_kernel", ([&] {
    paired_search_backward_kernel<scalar_t,0><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          stride0, ps, pt, dilation, patch_offset, reflect_bounds,
          nW0, ftrs_per_thread);
    }));
  }else if (dist_type == 1){ // l2
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_search_backward_kernel", ([&] {
    paired_search_backward_kernel<scalar_t,1><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          stride0, ps, pt, dilation, patch_offset, reflect_bounds,
          nW0, ftrs_per_thread);
    }));
  }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");    }


}



/****************************

       Backward Bilinear-2d

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_search_bilin2d_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame0,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame1,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_flow,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> flow,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_dists,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_inds,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds,
    int stride0, int ps, int dilation, int patch_offset, bool reflect_bounds,
    int nW0, int ftrs_per_thread) {

  // -- shape --
  int nbatch = grad_dists.size(0);
  int Q = grad_dists.size(2);
  int K =  grad_dists.size(3);
  int HD_frame = frame0.size(1);
  int HD_flow = grad_flow.size(1);
  int HD_search = inds.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int HD = max(HD_frame,HD_flow);

  // -- fwd decl registers --
  int ref_patch[2];
  scalar_t prop_patch[2];
  int ref[2];
  scalar_t prop[2];
  int prop_i[2];
  bool valid_ref[3];
  bool valid_prop[3];
  int qindex,qindex_tmp;
  bool valid;
  scalar_t weight,pix0,pix1,pix;
  scalar_t iweight[2];
  int iftr;
  // int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- location to fill --
  int i0 = blockIdx.x*blockDim.x+threadIdx.x;
  int i1 = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/nbatch;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int ihead_sr = ihead % HD_search;
  int ibatch = (blockIdx.z-ihead*nbatch);

  // -- feature chunk --
  int ftr_start = threadIdx.z * ftrs_per_thread;
  int ftr_end = min(F,ftr_start + ftrs_per_thread);

  // -- each region --
  if ((i0 < Q) && (i1 < K)){

    // -- full-resolution video query index --
    qindex = i0;// + q_shift;

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qindex,qindex_tmp,stride0,nW0,H,W);

    // -- proposed location --
    prop_patch[0] = inds[ibatch][ihead_sr][i0][i1][0];
    prop_patch[1] = inds[ibatch][ihead_sr][i0][i1][1];
    prop_patch[2] = inds[ibatch][ihead_sr][i0][i1][2];
    weight = grad_dists[ibatch][ihead_sr][i0][i1];
    iweight[0] = grad_inds[ibatch][ihead_sr][i0][i1][0];
    iweight[1] = grad_inds[ibatch][ihead_sr][i0][i1][1];

    // -- update frames --
    update_bwd_bilin2d_patch_2d<scalar_t,DIST_TYPE>(
                     grad_frame0[ibatch][ihead_fr],
                     grad_frame1[ibatch][ihead_fr],
                     frame0[ibatch][ihead_fr],
                     frame1[ibatch][ihead_fr],
                     weight,ref_patch,prop_patch,
                     ps,dilation,reflect_bounds,
                     patch_offset,iftr,ftr_start,ftr_end,
                     ref,prop,prop_i,
                     valid_ref,valid_prop,valid,
                     H,W,pix0,pix1,pix,i1);

    // -- update flows --
    if (ftr_start == 0){
      scalar_t wi = ref[1] + flow[ibatch][ihead_fl][0][ref[0]][ref[1]];
      scalar_t hi = ref[0] + flow[ibatch][ihead_fl][1][ref[0]][ref[1]];
      // int sW = ((wi >= 0) and (wi < W-1)) ? 1 : -1;
      // int sH = ((hi >= 0) and (hi < H-1)) ? 1 : -1;
      int sW = ((wi >= 0) and (wi < W)) ? 1 : -1; // untested move from "W-1" to "W"
      int sH = ((hi >= 0) and (hi < H)) ? 1 : -1;
      atomicAdd(&(grad_flow[ibatch][ihead_fl][0][ref[0]][ref[1]]),sW*iweight[1]);
      atomicAdd(&(grad_flow[ibatch][ihead_fl][1][ref[0]][ref[1]]),sH*iweight[0]);
    }

  }
}

void paired_search_bilin2d_backward_cuda(
    torch::Tensor grad_frame0, torch::Tensor grad_frame1,
    torch::Tensor grad_flow,
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow,
    const torch::Tensor grad_dists, const torch::Tensor grad_inds,
    const torch::Tensor inds,
    int stride0, int ps, int dilation, bool reflect_bounds,
    int patch_offset, int dist_type) {

  // -- unpack --
  int B = frame0.size(0);
  int HD_frame = frame0.size(1);
  int HD_flow = grad_dists.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  int nqueries = inds.size(2);
  int K = inds.size(3);
  // assert(pt == 1);
  int HD = max(HD_frame,HD_flow);

  // -- launch parameters --
  int nbatch = grad_dists.size(0);
  int nq = grad_dists.size(2);
  int k = grad_dists.size(3);
  int ftr_threads = min(1,F);
  dim3 threadsPerBlock(448,2,ftr_threads);
  dim3 blocksPerGrid(1, 1, nbatch*HD);
  blocksPerGrid.x = ceil(double(nq)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(k)/double(threadsPerBlock.y));
  int ftrs_per_thread = (F-1)/ftr_threads+1;
  int nW0 = (W-1)/stride0+1;

  // -- shared --
  // int psHalf = ps/2;
  // int adj = use_adj ? psHalf : 0;
  // int patch_offset = adj - psHalf;
  // int patch_offset = psHalf - adj;

  // -- launch kernel --
  if (dist_type == 0){ // prod
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                               "paired_search_bilin2d_backward_kernel", ([&] {
    paired_search_bilin2d_backward_kernel<scalar_t,0><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          grad_inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          stride0, ps, dilation, patch_offset, reflect_bounds, nW0, ftrs_per_thread);
    }));
  }else if (dist_type == 1){ // l2
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                               "paired_search_bilin2d_backward_kernel", ([&] {
    paired_search_bilin2d_backward_kernel<scalar_t,1><<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          flow.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          grad_inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          stride0, ps, dilation, patch_offset, reflect_bounds, nW0, ftrs_per_thread);
    }));
  }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");    }


}


