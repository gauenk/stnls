#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstddef>
#include "shared_kernel.cu"
using namespace at;

/****************************

       Forward Pass

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void refinement_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> vid0,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> vid1,
    const torch::PackedTensorAccessor32<int,5,torch::RestrictPtrTraits> qinds,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<int,7,torch::RestrictPtrTraits> inds,
    int wr_h, int wr_w, int ws_h2, int ws_w2,
    int ps, int pt, int k, int stride0, int stride1, int dilation,
    int q_shift, int nH0, int nW0, int nHW0, bool reflect_bounds, bool full_ws,
    bool use_adj, int off_H0, int off_W0, int off_H1, int off_W1,
    int q_per_thread, int k_per_thread, int wr_h_per_thread, int wr_w_per_thread){

  // -- unpack shapes --
  int HD = vid0.size(1);
  int T = vid0.size(2);
  int C = vid0.size(3);
  int H = vid0.size(4);
  int W = vid0.size(5);
  int Q = dists.size(2);
  int K = qinds.size(3);

  // -- invalid constant --
  scalar_t invalid = (scalar_t)__int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    invalid = -invalid;
  }

  // -- search region offsets --
  int psHalf = (ps)/2;
  int wrHalf_h = (wr_h)/2;
  int wrHalf_w = (wr_w)/2;
  int wrOff_h = wrHalf_h;
  int wrOff_w = wrHalf_w;
  int wrMax_h = stride1*(wr_h-1-wrOff_h);
  int wrMax_w = stride1*(wr_w-1-wrOff_w);
  int wrMin_h = -stride1 * wrOff_h;
  int wrMin_w = -stride1 * wrOff_h;
  int adj = use_adj ? psHalf : 0;

  // -- cuda index --
  int ibatch = blockIdx.x;
  int ihead = blockIdx.y;
  int q_start = blockIdx.z*q_per_thread;
  int qi,si,wh,ww;
  int qindex,qindex_tmp;

  // -- fwd decls --
  int prop_center[2];
  int prop_patch[3];
  int prop_pix[3];
  int ref_patch[3];
  int ref_pix[3];
  bool valid;
  bool valid_prop[4];
  bool valid_ref[4];
  scalar_t dist,pix0,pix1,_dist;

  // int ti,hi,wi;
  // int n_ti,n_hi,n_wi;
  // int vH,vW,vT,nH,nW,nT;
  // bool valid_anchor,valid_n,valid;
  // bool vvalid_t,vvalid_h,vvalid_w,vvalid;
  // bool nvalid_t,nvalid_h,nvalid_w,nvalid;

  // -- cleaner code --
  int center_offsets[4] = {off_H0,off_W0,off_H1,off_W1};
  int patch_offset = psHalf + adj;


  for (int q_index = 0; q_index < q_per_thread; q_index++){

    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }
    qindex = qi + q_shift;

    // -- pixel location from query index --
    get_pixel_loc(ref_patch,qindex,qindex_tmp,stride0,nW0,nHW0,H,W);

    // -- check bounds of pixel location --
    check_bounds(valid_ref[3],ref_patch,T,H,W);

    // -- search region offsets --
    set_search_offsets(wrOff_h,wrOff_w, ref_patch[1], ref_patch[2],
                       stride1, wrHalf_h, wrHalf_w, wrMax_h, wrMax_w, H, W, full_ws);

    // -- [unused] set search bounds for [optionally] expanded region --
    // set_search_minmax(wrMax_h, wrMin_h, wrOff_h, wr_h, stride1, full_ws);
    // set_search_minmax(wrMax_w, wrMin_w, wrOff_w, wr_w, stride1, full_ws);

    // ---------------------------------------
    //     for each neighbor in k_search
    // ---------------------------------------
    for(int _si = 0; _si < k_per_thread; _si++){
      si = threadIdx.x + blockDim.x*_si;
      if (si >= K){ continue; }

      // -- unpack base -- 
      prop_patch[0] = qinds[ibatch][ihead][qi][si][0]; // no search
      prop_center[0] = qinds[ibatch][ihead][qi][si][1];
      prop_center[1] = qinds[ibatch][ihead][qi][si][2];

      // ---------------------------------------
      //     for each position to search
      // ---------------------------------------
      for(int _wh = 0; _wh < wr_h_per_thread; _wh++){
        wh = threadIdx.y + blockDim.y*_wh;
        if (wh >= wr_h){ continue; }

        for(int _ww = 0; _ww < wr_w_per_thread; _ww++){
          ww = threadIdx.z + blockDim.z*_ww;
          if (ww >= wr_w){ continue; }

          // --------------------
          //      init dists
          // --------------------
          dist = 0;

          // ----------------------
          //    spatial center
          // ----------------------
          prop_patch[1] = (prop_center[0]) + stride1 * (wh - wrOff_h);
          prop_patch[2] = (prop_center[1]) + stride1 * (ww - wrOff_w);

          // -- check bounds of pixel location --
          check_bounds(valid_prop[3],prop_patch,T,H,W);
          valid = valid_ref[3] && valid_prop[3];

          //  -- compute patch difference --
          if (valid){
            compute_dist<scalar_t,DIST_TYPE>(dist,
                         vid0[ibatch][ihead],vid1[ibatch][ihead],
                         ref_patch, prop_patch, 
                         ref_pix, prop_pix, valid_ref, valid_prop,
                         ps,pt,dilation,reflect_bounds,
                         patch_offset,center_offsets,invalid,
                         T,C,H,W,pix0,pix1,_dist);
          }

          // -- assignent --
          if (!valid){ dist = invalid; }
          dists[ibatch][ihead][qi][si][wh][ww] = dist;
          inds[ibatch][ihead][qi][si][wh][ww][0] = prop_patch[0];
          inds[ibatch][ihead][qi][si][wh][ww][1] = prop_patch[1];
          inds[ibatch][ihead][qi][si][wh][ww][2] = prop_patch[2];

        } //  ww
      } // wh
    } // si
  } // qi
} // fxn

void refinement_forward_cuda(
    const torch::Tensor vid0, const torch::Tensor vid1,
    const torch::Tensor qinds, torch::Tensor dists, torch::Tensor inds,
    int ws_h, int ws_w, int ps, int k, int dist_type, int stride0, int stride1,
    int dilation, int pt, int q_shift, bool reflect_bounds, bool full_ws,
    bool use_adj, int off_H0, int off_W0, int off_H1, int off_W1){

   // -- num threads --
   int nheads = dists.size(1);
   int nqueries = dists.size(2);
   int ksearch = inds.size(3);
   int wr_h = dists.size(4);
   int wr_w = dists.size(5);
   int ksearch_threads = std::min(ksearch,12);
   int wr_h_threads = std::min(wr_h,8);
   int wr_w_threads = std::min(wr_w,8);
   int k_per_thread = ((ksearch-1)/ksearch_threads)+1;
   int wr_h_per_thread = ((wr_h-1)/wr_h_threads) + 1;
   int wr_w_per_thread = ((wr_w-1)/wr_w_threads) + 1;
   dim3 nthreads(ksearch_threads,wr_h_threads,wr_w_threads);

   int batchsize = vid0.size(0);
   int rem_blocks = (65535-1)/nheads+1;
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   nquery_blocks = min(nquery_blocks,rem_blocks);
   q_per_thread = ((nqueries - 1) / nquery_blocks) + 1;
   dim3 nblocks(batchsize,nheads,nquery_blocks);

   // -- derived quantities --
   int H = vid0.size(4);
   int W = vid0.size(5);
   int nH0 = (H-1)/stride0+1;
   int nW0 = (W-1)/stride0+1;
   int nHW0 = nH0 * nW0;
   int ws_h2 = ws_h/2;
   int ws_w2 = ws_w/2;

   // launch kernel
   if (dist_type == 0){
     AT_DISPATCH_FLOATING_TYPES(vid0.type(),"refinement_forward_kernel", ([&] {
          refinement_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
          vid0.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          vid1.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          qinds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          dists.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,7,torch::RestrictPtrTraits>(),
          wr_h, wr_w, ws_h2, ws_w2, ps, pt, k, stride0, stride1, dilation,
          q_shift, nH0, nW0, nHW0, reflect_bounds, full_ws,
          use_adj, off_H0, off_W0, off_H1, off_W1,
          q_per_thread, k_per_thread, wr_h_per_thread, wr_w_per_thread);
        }));
   }else if (dist_type == 1){
     AT_DISPATCH_FLOATING_TYPES(vid0.type(),"refinement_forward_kernel", ([&] {
          refinement_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
          vid0.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          vid1.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          qinds.packed_accessor32<int,5,torch::RestrictPtrTraits>(),
          dists.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<int,7,torch::RestrictPtrTraits>(),
          wr_h, wr_w, ws_h2, ws_w2, ps, pt, k, stride0, stride1, dilation,
          q_shift, nH0, nW0, nHW0, reflect_bounds, full_ws,
          use_adj, off_H0, off_W0, off_H1, off_W1,
          q_per_thread, k_per_thread, wr_h_per_thread, wr_w_per_thread);
        }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}


