#include "hip/hip_runtime.h"

// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstddef>
#include "paired_details.cu"
// #include "nls_bilin2d.cu"

using namespace at;


/****************************

       Forward Pass

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_refine_int_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> flow,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<int,7,torch::RestrictPtrTraits> inds,
    int ws, int ps, int stride0, int stride1, int dilation,
    bool reflect_bounds, bool full_ws, int patch_offset,
    int off_Hq, int off_Wq, int q_per_thread, int wr_per_thread){

  // -- unpack shape --
  int B = frame0.size(0);
  int HD_frame = frame0.size(1);
  int HD_flow = flow.size(1);
  int HD_search = inds.size(1);
  int C = frame0.size(2);
  int qH = frame0.size(3);
  int qW = frame0.size(4);
  int kH = frame1.size(3);
  int kW = frame1.size(4);
  int Q = dists.size(2);
  int HD = max(HD_frame,HD_flow);
  int wr = dists.size(5); // 4 or 5

  // -- invalid constant --
  scalar_t invalid = (scalar_t)__int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    invalid = -invalid;
  }

  // -- offsets --
  int offs[2];
  offs[0] = off_Hq;
  offs[1] = off_Wq;

  // -- search region offsets --
  int wrHalf = (wr-1)/2;
  int wrOff_h,wrOff_w;

  // -- cuda index --
  int ihead = blockIdx.z/B;
  int ibatch = (blockIdx.z-ihead*B) % B;
  int si = blockIdx.y;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int ihead_sr = ihead % HD_search;
  int q_start = blockIdx.x*q_per_thread;
  int qi,wr_i,wr_j;

  // decls
  int ref_patch[2];
  int prop_patch[2];
  int frame_anchor[2];
  int ref_pix[2];
  int prop_pix[2];
  bool valid;
  bool valid_ref_patch,valid_prop_patch;
  bool valid_ref[3];
  bool valid_prop[3];

  // -- indexing --
  scalar_t dist;

  for (int q_index = 0; q_index < q_per_thread; q_index++){


    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qi,stride0,qH,qW);
    int nh = ref_patch[0]/stride0;
    int nw = ref_patch[1]/stride0;
    check_bounds_2d(valid_ref_patch,ref_patch,qH,qW);

    // -- assign to reference --
    frame_anchor[0] = ref_patch[0] + flow[ibatch][ihead_fl][nh][nw][si][0];
    frame_anchor[1] = ref_patch[1] + flow[ibatch][ihead_fl][nh][nw][si][1];
    frame_anchor[0] = bounds(frame_anchor[0],kH);
    frame_anchor[1] = bounds(frame_anchor[1],kW);

    // -- search region offsets --
    set_search_offsets(wrOff_h, wrOff_w,
                       frame_anchor[0], frame_anchor[1], stride1,
                       wrHalf, wr, kH, kW, full_ws);

    // ---------------------------------------
    //          spatial searching
    // ---------------------------------------

    // -- search across space --
    for (int _xi = 0; _xi < wr_per_thread; _xi++){
      wr_i = threadIdx.x + blockDim.x*_xi;
      if (wr_i >= wr){ continue; }
      for (int _yi = 0; _yi < wr_per_thread; _yi++){
        wr_j = threadIdx.y + blockDim.y*_yi;
        if (wr_j >= wr){ continue; }

        // -- compute proposed location --
        prop_patch[0] = frame_anchor[0] + stride1 * (wr_i - wrOff_h);
        prop_patch[1] = frame_anchor[1] + stride1 * (wr_j - wrOff_w);
        check_bounds_2d(valid_prop_patch,prop_patch,kH,kW);
        valid = valid_ref_patch && valid_prop_patch;

        // -- init dist --
        dist = 0;

        //  -- compute patch difference --
        if (valid){

          compute_dist_2d<scalar_t,DIST_TYPE>(dist,
                       frame0[ibatch][ihead_fr],frame1[ibatch][ihead_fr],
                       ref_patch, prop_patch, 
                       ref_pix, prop_pix, valid_ref, valid_prop,
                       ps,dilation,reflect_bounds,
                       patch_offset,invalid,offs,C,qH,qW,kH,kW);

        }

        // -- assignent --
        if (!valid){ dist = invalid; }
        dists[ibatch][ihead_sr][qi][si][wr_i][wr_j] = dist;
        inds[ibatch][ihead_sr][qi][si][wr_i][wr_j][0] = prop_patch[0]-ref_patch[0];
        inds[ibatch][ihead_sr][qi][si][wr_i][wr_j][1] = prop_patch[1]-ref_patch[1];
          
      }
    }
  }
}

void paired_refine_int_forward_cuda(
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow, torch::Tensor dists, torch::Tensor inds,
    int ws, int ps, int stride0, int stride1, int dilation,
    bool restricted_radius, bool reflect_bounds, bool full_ws,
    int patch_offset, int off_Hq, int off_Wq, int dist_type){

   // -- derived quantities --
   int B = frame0.size(0);
   int HD_frame = frame0.size(1);
   int HD_flow = flow.size(1);
   int H = frame0.size(3);
   int W = frame0.size(4);
   int S = flow.size(4);
   int HD = max(HD_frame,HD_flow);

   // -- threads --
   int nqueries = dists.size(2);
   int wr = dists.size(4);
   int wr_threads = std::min(wr,25);
   int wr_per_thread = ((wr-1)/wr_threads) + 1;
   dim3 nthreads(wr_threads,wr_threads);

   // -- nblocks --
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   dim3 nblocks(nquery_blocks,S,B*HD);

   // -- launch kernel --
   if (dist_type == 0){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_refine_int_forward_kernel", ([&] {
       paired_refine_int_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<int,7,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, off_Hq, off_Wq, q_per_thread, wr_per_thread);
          }));
   }else if(dist_type == 1){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),"paired_refine_int_forward_kernel", ([&] {
       paired_refine_int_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<int,7,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, off_Hq, off_Wq, q_per_thread, wr_per_thread);
          }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}


/**********************************

      Forward Pass (Bilin2d)

**********************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_refine_bilin2d_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> flow,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> dists,
    torch::PackedTensorAccessor32<scalar_t,7,torch::RestrictPtrTraits> inds,
    torch::PackedTensorAccessor32<int,6,torch::RestrictPtrTraits> kselect,
    int ws, int ps, int stride0, float _stride1, int dilation,
    bool reflect_bounds, bool full_ws, int patch_offset,
    int off_Hq, int off_Wq, int q_per_thread, int wr_per_thread){

  // -- unpack shape --
  int B = frame0.size(0);
  int HD_frame = frame0.size(1);
  int HD_flow = flow.size(1);
  int HD_search = inds.size(1);
  int C = frame0.size(2);
  int qH = frame0.size(3);
  int qW = frame0.size(4);
  int kH = frame1.size(3);
  int kW = frame1.size(4);
  int Q = dists.size(2);
  int HD = max(HD_frame,HD_flow);
  scalar_t stride1 = static_cast<scalar_t>(_stride1);
  int wr = dists.size(4);

  // -- invalid constant --
  scalar_t invalid = (scalar_t)__int_as_float(0x7f800000);
  if(DIST_TYPE == 0){ // prod
    invalid = -invalid;
  }

  // -- offsets --
  int offs[2];
  offs[0] = off_Hq;
  offs[1] = off_Wq;

  // -- search region offsets --
  // int psHalf = (ps)/2;
  // int wsHalf_h = (ws_h)/2;
  // int wsHalf_w = (ws_w)/2;
  // int wsMax_h = stride1*(ws_h-1-wsHalf_h);
  // int wsMax_w = stride1*(ws_w-1-wsHalf_w);
  // int adj = use_adj ? psHalf : 0;

  // int wsHalf_h = (ws_h-1)/2;
  // int wsHalf_w = (ws_w-1)/2;
  // int wsOff_h,wsOff_w;
  scalar_t wrHalf = trunc((wr-1)/2);
  scalar_t wrOff_h,wrOff_w;

  // -- cuda index --
  int ihead = blockIdx.z/B;
  int ibatch = (blockIdx.z-ihead*B) % B;
  int si = blockIdx.y;
  // int ibatch = blockIdx.y;
  // int ihead = blockIdx.z;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int ihead_sr = ihead % HD_search;
  int q_start = blockIdx.x*q_per_thread;
  int qi,wr_i,wr_j;

  // decls
  int ref_patch[2];
  scalar_t prop_patch[2];
  scalar_t frame_anchor[2];
  int ref_pix[2];
  scalar_t prop_pix[2];
  // int prop_i[2];
  bool valid;
  bool valid_ref_patch,valid_prop_patch;
  bool valid_ref[3];
  bool valid_prop[3];

  // -- indexing --
  scalar_t dist,pix0,pix1;

  for (int q_index = 0; q_index < q_per_thread; q_index++){


    //---------------------------
    //       Anchor Pixel
    //---------------------------

    // -- block start --
    qi = q_start + q_index;
    if (qi >= Q){ continue; }

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qi,stride0,qH,qW);
    check_bounds_2d(valid_ref_patch,ref_patch,qH,qW);
    int nh = ref_patch[0]/stride0;
    int nw = ref_patch[1]/stride0;

    // -- compute frame offsets with flow --
    frame_anchor[0] = ref_patch[0] + flow[ibatch][ihead_fl][nh][nw][si][0];
    frame_anchor[1] = ref_patch[1] + flow[ibatch][ihead_fl][nh][nw][si][1];
    // frame_anchor[0] = ref_patch[0]+flow[ibatch][ihead_fl][1][nh][nw];
    // frame_anchor[1] = ref_patch[1]+flow[ibatch][ihead_fl][0][nh][nw];
    frame_anchor[0] = bounds(frame_anchor[0],kH);
    frame_anchor[1] = bounds(frame_anchor[1],kW);

    // -- search region offsets --
    set_search_offsets(wrOff_h, wrOff_w,
                       frame_anchor[0], frame_anchor[1], stride1,
                       wrHalf, wr, kH, kW, full_ws);

    // ---------------------------------------
    //          spatial searching
    // ---------------------------------------

    // -- search across space --
    for (int _xi = 0; _xi < wr_per_thread; _xi++){
      wr_i = threadIdx.x + blockDim.x*_xi;
      if (wr_i >= wr){ continue; }
      for (int _yi = 0; _yi < wr_per_thread; _yi++){
        wr_j = threadIdx.y + blockDim.y*_yi;
        if (wr_j >= wr){ continue; }

        // -- compute proposed location --
        prop_patch[0] = frame_anchor[0] + stride1 * (wr_i - wrOff_h);
        prop_patch[1] = frame_anchor[1] + stride1 * (wr_j - wrOff_w);
        check_bounds_2d<scalar_t>(valid_prop_patch,prop_patch,kH,kW);
        valid = valid_ref_patch && valid_prop_patch;


        // -- init dist --
        dist = 0;
        // Z = 0;

        //  -- compute patch difference --
        if (valid){
          compute_dist_bilin2d_2d<scalar_t,DIST_TYPE>(dist,
                       frame0[ibatch][ihead_fr],frame1[ibatch][ihead_fr],
                       ref_patch, prop_patch, ref_pix, prop_pix,// prop_i,
                       valid_ref, valid_prop, ps,dilation,reflect_bounds,
                       patch_offset,invalid,offs,C,qH,qW,kH,kW);
        }


        // -- assignent --
        if (!valid){ dist = invalid; }
        dists[ibatch][ihead_sr][qi][si][wr_i][wr_j] = dist;
        inds[ibatch][ihead_sr][qi][si][wr_i][wr_j][0] = prop_patch[0]-ref_patch[0];
        inds[ibatch][ihead_sr][qi][si][wr_i][wr_j][1] = prop_patch[1]-ref_patch[1];
        kselect[ibatch][ihead][qi][si][wr_i][wr_j] = si;
          
      }
    }
  }
}

void paired_refine_bilin2d_forward_cuda(
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow,
    torch::Tensor dists, torch::Tensor inds, torch::Tensor kselect,
    int ws, int ps, int stride0, float stride1, int dilation,
    bool restricted_radius, bool reflect_bounds, bool full_ws,
    int patch_offset, int off_Hq, int off_Wq, int dist_type){

   // -- derived quantities --
   int B = frame0.size(0);
   int HD_frame = frame0.size(1);
   int HD_flow = flow.size(1);
   int H = frame0.size(3);
   int W = frame0.size(4);
   // int nH0 = (H-1)/stride0+1;
   int HD = max(HD_frame,HD_flow);
   int S = flow.size(4);

   // -- threads --
   int nqueries = dists.size(2);
   int wr = dists.size(4);
   int wr_threads = std::min(wr,15);
   int wr_per_thread = ((wr-1)/wr_threads) + 1;
   dim3 nthreads(wr_threads,wr_threads);

   // -- nblocks --
   int q_per_thread = 2;
   int nquery_blocks = ((nqueries - 1) / q_per_thread) + 1;
   dim3 nblocks(nquery_blocks,S,B*HD);

   // -- share --
   // int psHalf = ps/2;
   // int adj = use_adj ? psHalf : 0;
   // // int patch_offset = adj - psHalf;
   // int patch_offset = adj - psHalf;

   // -- viz --
   // fprintf(stdout,"wr_h,wr_w: %d,%d\n",wr_h,wr_w);
   // fprintf(stdout,"nquery_blocks,B,HD: %d,%d,%d\n",nquery_blocks,B,HD);

   // launch kernel
   if (dist_type == 0){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                                  "paired_refine_bilin2d_forward_kernel", ([&] {
       paired_refine_bilin2d_forward_kernel<scalar_t,0><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<scalar_t,7,torch::RestrictPtrTraits>(),
            kselect.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, off_Hq, off_Wq, q_per_thread, wr_per_thread);
          }));
   }else if(dist_type == 1){
       AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                                  "paired_refine_bilin2d_forward_kernel", ([&] {
       paired_refine_bilin2d_forward_kernel<scalar_t,1><<<nblocks, nthreads>>>(
            frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
            flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            dists.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
            inds.packed_accessor32<scalar_t,7,torch::RestrictPtrTraits>(),
            kselect.packed_accessor32<int,6,torch::RestrictPtrTraits>(),
            ws, ps, stride0, stride1, dilation, reflect_bounds, full_ws,
            patch_offset, off_Hq, off_Wq, q_per_thread, wr_per_thread);
          }));
   }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");
   }
}



/****************************

       Backward Bilinear-2d

****************************/

template <typename scalar_t, int DIST_TYPE>
__global__ void paired_refine_vidflows_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame0,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_frame1,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> grad_flow,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame0,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> frame1,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> flow,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_dists,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_inds,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<int,4,torch::RestrictPtrTraits> kselect,
    int stride0, int ps, int dilation, int patch_offset, bool reflect_bounds,
    int off_Hq, int off_Wq) {

  // -- shape --
  int nbatch = grad_dists.size(0);
  int Q = grad_dists.size(2);
  int K =  grad_dists.size(3);
  int HD_frame = frame0.size(1);
  int HD_flow = grad_flow.size(1);
  int HD_search = inds.size(1);
  int F = frame0.size(2);
  int qH = frame0.size(3);
  int qW = frame0.size(4);
  int kH = frame1.size(3);
  int kW = frame1.size(4);
  int HD = max(HD_frame,HD_flow);

  // -- fwd decl registers --
  int ref_patch[2];
  scalar_t prop_patch[2];
  // int ref[2];
  // scalar_t prop[2];
  // int prop_i[2];
  bool valid_ref[3];
  bool valid_prop[3];
  bool valid;
  scalar_t weight;
  scalar_t iweight[2];
  // int center_offsets[4] = {off_H0,off_H1,off_W0,off_W1};

  // -- offsets --
  int offs[2];
  offs[0] = off_Hq;
  offs[1] = off_Wq;

  // -- location to fill --
  int qi = blockIdx.x*blockDim.x+threadIdx.x;
  int ki = blockIdx.y*blockDim.y+threadIdx.y;
  int ihead = blockIdx.z/nbatch;
  int ihead_fr = ihead % HD_frame;
  int ihead_fl = ihead % HD_flow;
  int ihead_sr = ihead % HD_search;
  int ibatch = (blockIdx.z-ihead*nbatch);

  // -- feature chunk --
  // int ftr_start = 0;//threadIdx.z * ftrs_per_thread;
  // int ftr_end = F;//min(F,ftr_start + ftrs_per_thread);

  // -- each region --
  if ((qi < Q) && (ki < K)){

    // -- pixel location from query index --
    get_pixel_loc_2d(ref_patch,qi,stride0,qH,qW);
    int nh = ref_patch[0]/stride0;
    int nw = ref_patch[1]/stride0;

    // -- accumulate optical flow update --
    scalar_t acc_dFlows[8];
  #pragma unroll
    for (int _idx=0; _idx < 8; _idx++){
      acc_dFlows[_idx] = static_cast<scalar_t>(0);
    }

    // -- proposed location --
    prop_patch[0] = ref_patch[0] + inds[ibatch][ihead_sr][qi][ki][0];
    prop_patch[1] = ref_patch[1] + inds[ibatch][ihead_sr][qi][ki][1];
    prop_patch[0] = bounds(prop_patch[0],kH);
    prop_patch[1] = bounds(prop_patch[1],kW);

    weight = grad_dists[ibatch][ihead_sr][qi][ki];
    iweight[0] = grad_inds[ibatch][ihead_sr][qi][ki][0];
    iweight[1] = grad_inds[ibatch][ihead_sr][qi][ki][1];
    int kj = kselect[ibatch][ihead][qi][ki];

    // -- update frames --
    update_bwd_bilin2d_patch_2d<scalar_t,DIST_TYPE>(
                     grad_frame0[ibatch][ihead_fr],grad_frame1[ibatch][ihead_fr],
                     frame0[ibatch][ihead_fr],frame1[ibatch][ihead_fr],
                     acc_dFlows,weight,ref_patch,prop_patch,
                     ps,dilation,reflect_bounds,patch_offset,
                     valid_ref,valid_prop,valid,offs,qH,qW,kH,kW);


    // -- update grad_flow from grad_dists,vid0,vid1 --
    scalar_t hi = ref_patch[0] + flow[ibatch][ihead_fl][nh][nw][kj][0];
    scalar_t wi = ref_patch[1] + flow[ibatch][ihead_fl][nh][nw][kj][1];
    int signH = ((hi >= 0) and (hi <= (kH-1))) ? 1 : -1;
    int signW = ((wi >= 0) and (wi <= (kW-1))) ? 1 : -1;
    bwd_flow_assign_v3(acc_dFlows,signH,signW,
                       grad_flow[ibatch][ihead_fl][nh][nw][kj]);

    // -- update flows --
    atomicAdd(&(grad_flow[ibatch][ihead_fl][nh][nw][kj][0]),signH*iweight[0]);
    atomicAdd(&(grad_flow[ibatch][ihead_fl][nh][nw][kj][1]),signW*iweight[1]);

  }
}

void paired_refine_vidflows_backward_cuda(
    torch::Tensor grad_frame0, torch::Tensor grad_frame1,
    torch::Tensor grad_flow,
    const torch::Tensor frame0, const torch::Tensor frame1,
    const torch::Tensor flow,
    const torch::Tensor grad_dists, const torch::Tensor grad_inds,
    const torch::Tensor inds, const torch::Tensor kselect,
    int stride0, int ps, int dilation, bool reflect_bounds,
    int patch_offset, int off_Hq, int off_Wq, int dist_type) {

  // -- unpack --
  int HD_frame = frame0.size(1);
  int HD_flow = grad_dists.size(1);
  int F = frame0.size(2);
  int H = frame0.size(3);
  int W = frame0.size(4);
  // int K = inds.size(3);
  // assert(pt == 1);
  int HD = max(HD_frame,HD_flow);

  // -- launch parameters --
  int B = grad_dists.size(0);
  int Q = grad_dists.size(2);
  int K = grad_dists.size(3);
  dim3 threadsPerBlock(288,2);
  dim3 blocksPerGrid(1, 1, B*HD);
  blocksPerGrid.x = ceil(double(Q)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(K)/double(threadsPerBlock.y));

  // -- shared --
  // int psHalf = ps/2;
  // int adj = use_adj ? psHalf : 0;
  // int patch_offset = adj - psHalf;
  // int patch_offset = psHalf - adj;

  // -- launch kernel --
  if (dist_type == 0){ // prod
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                               "paired_refine_vidflows_backward_kernel", ([&] {
    paired_refine_vidflows_backward_kernel<scalar_t,0>
      <<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          grad_inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          kselect.packed_accessor32<int,4,torch::RestrictPtrTraits>(),
          stride0, ps, dilation, patch_offset, reflect_bounds, off_Hq, off_Wq);
    }));
  }else if (dist_type == 1){ // l2
    AT_DISPATCH_FLOATING_TYPES(frame0.type(),
                               "paired_refine_vidflows_backward_kernel", ([&] {
    paired_refine_vidflows_backward_kernel<scalar_t,1>
      <<<blocksPerGrid, threadsPerBlock>>>(
          grad_frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          grad_flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          frame0.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          frame1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          flow.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
          grad_dists.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          grad_inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          inds.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
          kselect.packed_accessor32<int,4,torch::RestrictPtrTraits>(),
          stride0, ps, dilation, patch_offset, reflect_bounds, off_Hq, off_Wq);
    }));
  }else{
     throw std::invalid_argument("Uknown distance type. Must be 0 (product) or 1 (l2)");    }


}


